#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>


/*const int fila;
const int columna;
*/
int vidas = 5;



// Esta función genera una semilla aleatoria basada en la hora actual.
unsigned int generate_seed() {
    time_t t;
    time(&t);
    return (unsigned int)t % 100000;
}

__global__ void mover_candy(int* mat, int n, int m, int dir,int cordx,int cordy) {
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int fila = threadIdx.y + blockDim.y * blockIdx.y;
    int posicion = fila * n + col;
    int caramelo = cordy * n + cordx;
    if (posicion == caramelo) {
        switch (dir) {
        case 1://Arriba
            if (cordy != 0) {
                int temp=mat[caramelo];
                mat[caramelo] = mat[caramelo - n];
                mat[caramelo - n] = temp;
            }
            break;
        case 2://Abajo
            if (cordy != m) {
                int temp = mat[caramelo];
                mat[caramelo] = mat[caramelo + n];
                mat[caramelo + n] = temp;
            }
            break;
        case 3://Derecha
            if (cordy != n) {
                int temp = mat[caramelo];
                mat[caramelo] = mat[caramelo + 1];
                mat[caramelo + 1] = temp;
            }
            break;
        case 4://Izquierda
            if (cordy != 0) {
                int temp = mat[caramelo];
                mat[caramelo] = mat[caramelo - 1];
                mat[caramelo - 1] = temp;
            }
            break;
        default:
            break;
        }
    }
    
}

// Esta función genera una matriz aleatoria de números enteros entre "lim_inf" y "lim_sup".
__global__ void random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    // Verificar si el hilo se encuentra dentro de los límites de la matriz
    if (idx < n && idy < m) {
        // Inicializar el generador de números aleatorios
        hiprand_init(ale, idx * m + idy, 0, &state[idx * m + idy]);
        // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
        int val = hiprand(&state[idx * m + idy]) % lim_sup + lim_inf;
        // Asignar el valor aleatorio a la matriz
        mat[idx * m + idy] = val;
    }
}


void create_random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup) {
    int* d_mat;
    hipMalloc(&d_mat, n * m * sizeof(int));

    hiprandState* d_state;
    hipMalloc(&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    random_matrix << <num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);

    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_state);
}



void imprimir(int* matriz, int n, int m) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", matriz[i * m + j]);
        }
        printf("\n");
    }
}


int generarNumAleatorio(int hasta) {
    
    
    srand(generate_seed());
    int ale = rand() % (hasta + 1);
    return ale;
}

int main()
{

    int modo; //automático o manual
    int dificultad; //dificultad del juego
    int n; // número de filas
    int m; // número de columnas
    printf("Bienvenido a Cundio Crack\n");
    printf("Introduce el modo de juego con el que quieres jugar: \n 1. Automatico \n 2. Manual \n");
    scanf("%d", &modo);
    printf("Introduce la dificultad con la que quieres jugar: \n 1. Facil \n 2. Normal \n");
    scanf("%d", &dificultad);
    printf("Introduce el numero de filas que quieres que tenga el tablero: \n");
    scanf("%d", &n);
    printf("Introduce el numero de columnas que quieres que tenga el tablero: \n");
    scanf("%d", &m);
    int lim_inf = 1; // valor mínimo
    int lim_sup = 6; // valor máximo
    if (dificultad == 1) {
         lim_sup = 4; // valor máximo
    }
    
    int* mat = (int*)malloc(n * m * sizeof(int)); // matriz aleatoria

    create_random_matrix(mat, n, m, lim_inf, lim_sup);


    //AQUI                                                                                  //<--ESTO HAY QUE VER COMO HACERLO EFICIENTE
    dim3 dimBlock(n,m);
    dim3 dimGrid(1);

    int cordx=-1;
    int cordy=-1;
    int dir=-1;

    while (vidas > 0) {
        imprimir(mat, n, m);
        do {
            if (modo == 2) {
                printf("Introduce la cordenada X del caramelo que quieres mover\n");
                scanf("%d", &cordx);
                printf("Introduce la cordenada Y del caramelo que quieres mover\n");
                scanf("%d", &cordy);
                printf("Introduce la dirección hacia la que quieres mover el caramelo deseado: \n 1 Arriba\n 2 Abajo\n 3 Derecha \n 4 Izquierda\n");
                scanf("%d", &dir);

            }
            else {                                                              //ESTA MIERDA ESTA MAL, REVISAR A VER COMO SERÍA
                srand(time(NULL));
                cordx = generarNumAleatorio(n);
                srand(time(NULL));
                cordy = generarNumAleatorio(m);
                srand(time(NULL));
                dir = generarNumAleatorio(4) + 1;
                printf("%d\n", cordx);
                printf("%d\n", cordy);
                printf("%d\n\n", dir);
            }
        } while (cordx > n && cordy > m && cordx < 0 && cordy < 0 && 0 > dir && dir > 5);

        int* d_mat;
        hipMalloc(&d_mat, n * m * sizeof(int));
        hipMemcpy(d_mat, mat, n * m * sizeof(int), hipMemcpyHostToDevice);

        mover_candy << <dimGrid, dimBlock >> > (d_mat, n, m, dir, cordx, cordy);

        hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_mat);

        vidas -= 1;
    }

   

    free(mat);

    return 0;
}