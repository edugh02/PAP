#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>


/*const int fila;
const int columna;
*/
int vidas = 5;
 //y filas, x columnas


// Esta función genera una semilla aleatoria basada en la hora actual.
unsigned int generate_seed() {
    time_t t;
    time(&t);
    return (unsigned int)t % 100000;
}

bool esta_o_no(int* vector, int m, int n, int pos) {
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == pos) {
            return true;
        }
    }
    return false;
}

int primer_vacio(int* vector, int n, int m) {
    int x = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == -1) {
            x = i;
            i = n * m;
        }
    }
    return x;
}


void ver_candy(int* mat, int n, int m,int cordx,int cordy, int* vector,int elemento) {
    int caramelo = cordy * n + cordx;
    printf("\nEl caramelo esta en la posicion: %d\n", caramelo);
    if ( !esta_o_no(vector,m,n,caramelo)&& mat[caramelo]==elemento) {
        int pos=primer_vacio(vector,n,m);
        printf("\nposicion del vector siguiente: %d \n", pos);
        vector[pos] = caramelo;
        if (cordy != 0) {
            ver_candy(mat, n, m, cordx, cordy - 1, vector,elemento);
        }
        if (cordy != n) {
            ver_candy(mat, n, m, cordx, cordy + 1, vector,elemento);
        }
        if (cordx != 0) {
            ver_candy(mat, n, m, cordx - 1, cordy, vector,elemento);
        }
        if (cordx != m) {
            ver_candy(mat, n, m, cordx + 1, cordy, vector,elemento);
        }
        
    }
    
    
}

// Esta función genera una matriz aleatoria de números enteros entre "lim_inf" y "lim_sup".
__global__ void random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    // Verificar si el hilo se encuentra dentro de los límites de la matriz
    if (idx < n && idy < m) {
        // Inicializar el generador de números aleatorios
        hiprand_init(ale, idx * m + idy, 0, &state[idx * m + idy]);
        // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
        int val = hiprand(&state[idx * m + idy]) % lim_sup + lim_inf;
        // Asignar el valor aleatorio a la matriz
        mat[idx * m + idy] = val;
    }
}


__global__ void fill(int* vec, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        vec[i] = -1;
    }
}

void crear_vector(int* posicionesVistas, int n, int m) {
    int* d_v;
    hipMalloc(&d_v, n * m * sizeof(int));

    // Definir la configuración del kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n * m + threadsPerBlock - 1) / threadsPerBlock;

    // Llamar al kernel
    fill << <blocksPerGrid, threadsPerBlock >> > (d_v, n * m);

    hipMemcpy(posicionesVistas, d_v, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_v);
}



void create_random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup) {
    int* d_mat;
    hipMalloc(&d_mat, n * m * sizeof(int));

    hiprandState* d_state;
    hipMalloc(&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    random_matrix << <num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);
    
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_state);
}



void imprimir(int* matriz, int n, int m) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", matriz[i * m + j]);
        }
        printf("\n");
    }
}


int generarNumAleatorio(int hasta) {
    srand(generate_seed());
    int ale = rand() % (hasta + 1);
    return ale;
}


int main()
{

    int modo; //automático o manual
    int dificultad; //dificultad del juego
    int n; // número de filas
    int m; // número de columnas
    printf("Bienvenido a Cundio Crack\n");
    printf("Introduce el modo de juego con el que quieres jugar: \n 1. Automatico \n 2. Manual \n");
    scanf("%d", &modo);
    printf("Introduce la dificultad con la que quieres jugar: \n 1. Facil \n 2. Normal \n");
    scanf("%d", &dificultad);
    printf("Introduce el numero de filas que quieres que tenga el tablero: \n");
    scanf("%d", &n);
    printf("Introduce el numero de columnas que quieres que tenga el tablero: \n");
    scanf("%d", &m);

    int lim_inf = 1; // valor mínimo
    int lim_sup = 6; // valor máximo
    if (dificultad == 1) {
         lim_sup = 4; // valor máximo
    }
    
    int* mat = (int*)malloc(n * m * sizeof(int)); // matriz aleatoria
    int* posicionesVistas = (int*)malloc(n * m * sizeof(int)); 
    crear_vector(posicionesVistas, n, m);
    create_random_matrix(mat, n, m, lim_inf, lim_sup);


    //AQUI                                                                                  //<--ESTO HAY QUE VER COMO HACERLO EFICIENTE
    dim3 dimBlock(n,m);
    dim3 dimGrid(1);

    int cordx=-1;
    int cordy=-1;
    int dir=-1;

    while (vidas > 0) {
        imprimir(mat, n, m);
        do {
            if (modo == 2) {
                printf("Introduce la cordenada X del caramelo que quieres mover\n");
                scanf("%d", &cordx);
                printf("Introduce la cordenada Y del caramelo que quieres mover\n");
                scanf("%d", &cordy);

            }
            else {                                                              //ESTA MIERDA ESTA MAL, REVISAR A VER COMO SERÍA
                srand(time(NULL));
                cordx = generarNumAleatorio(n);
                srand(time(NULL));
                cordy = generarNumAleatorio(m);
                printf("%d\n", cordx);
                printf("%d\n", cordy);
            }
        } while (cordx > n && cordy > m && cordx < 0 && cordy < 0);
        


        int elemento = mat[cordx * n + cordy];
        ver_candy(mat, n, m, cordx, cordy, posicionesVistas,elemento);
        for (int i = 0; i < n * m; ++i) {
            printf("%d ", posicionesVistas[i]);
        }
        printf("\n");

        vidas -= 1;
    }

   

    free(mat);

    return 0;
}