#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>


/*const int fila;
const int columna;
*/
int vidas = 5;
 //y filas, x columnas


// Esta función genera una semilla aleatoria basada en la hora actual.
unsigned int generate_seed() {
    time_t t;
    time(&t);
    return (unsigned int)t % 100000;
}

bool esta_en_vector(int* vector, int m, int n, int pos) {
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == pos) {
            return true;
        }
    }
    return false;
}

int primer_vacio(int* vector, int n, int m) {
    int x = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == -1) {
            x = i;
            i = n * m;
        }
    }
    return x;
}


void ver_candy(int* mat, int n, int m,int colum,int fila, int* vector,int elemento) {
    int caramelo = fila * m + colum;
    //printf("\nEl caramelo esta en la posicion: %d\n", caramelo);
    //printf("Está en el vector ya? %d\n",esta_en_vector(vector, m, n, caramelo));
    //printf("El elemento es igual? %d\n", mat[caramelo] == elemento);
    //printf("Elemento dentro %d \n", mat[caramelo]);

    if ( !esta_en_vector(vector,m,n,caramelo)&& mat[caramelo]==elemento) {
        int pos=primer_vacio(vector,n,m);
        printf("\nposicion del vector siguiente: %d \n", pos);
        vector[pos] = caramelo;
        if (fila != 0) {
            ver_candy(mat, n, m, colum, fila - 1, vector,elemento);
        }
        if (fila != n) {
            ver_candy(mat, n, m, colum, fila + 1, vector,elemento);
        }
        if (colum != 0) {
            ver_candy(mat, n, m, colum - 1, fila, vector,elemento);
        }
        if (colum != m) {
            ver_candy(mat, n, m, colum + 1, fila, vector,elemento);
        }
        
    }
    
    
}

// Esta función genera una matriz aleatoria de números enteros entre "lim_inf" y "lim_sup".
__global__ void random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    // Verificar si el hilo se encuentra dentro de los límites de la matriz
    if (idx < n && idy < m) {
        // Inicializar el generador de números aleatorios
        hiprand_init(ale, idx * m + idy, 0, &state[idx * m + idy]);
        // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
        int val = hiprand(&state[idx * m + idy]) % lim_sup + lim_inf;
        // Asignar el valor aleatorio a la matriz
        mat[idx * m + idy] = val;
    }
}



__global__ void eliminar_iguales_juntos(int* mat, int n, int m,int* vector) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos;

    for (int i = 0; i < n * m; i++) {
        if (vector[i] == idx * m + idy) {
            centinela = true; // El número está presente en el vector
            pos = i;
        } 
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idx * m + idy] = -1;
        vector[pos] = -1;
    }
}



void eliminar_elementos(int* matriz, int n, int m, int* vector) { //NO SE SI AQUI TMBN TENEMOS QUE ELIMINAR EL ELEMENTO DEL VECTOR DE POSICIONES 
    int* d_matriz;
    int* d_vector;
    int tamVector = n*m;

    // Alocar memoria para la matriz y el vector en la GPU
    hipMalloc(&d_matriz, n * m * sizeof(int));
    hipMalloc(&d_vector, tamVector * sizeof(int));

    // Copiar la matriz y el vector de la CPU a la GPU
    hipMemcpy(d_matriz, matriz, n * m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, vector, tamVector * sizeof(int), hipMemcpyHostToDevice);

    // Definir la configuración del kernel
    dim3 blockSize(1, 1);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);

    // Llamar al kernel
    eliminar_iguales_juntos << <gridSize, blockSize >> > (d_matriz, n, m, d_vector);

    // Copiar la matriz resultante de la GPU a la CPU
    hipMemcpy(matriz, d_matriz, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(vector, d_vector, n * m * sizeof(int), hipMemcpyDeviceToHost);
    
    // Liberar la memoria de la GPU
    hipFree(d_matriz);
    hipFree(d_vector);
}


__global__ void fill(int* vec, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        vec[i] = -1;
    }
}

void crear_vector(int* posicionesVistas, int n, int m) {
    int* d_v;
    hipMalloc(&d_v, n * m * sizeof(int));

    // Definir la configuración del kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n * m + threadsPerBlock - 1) / threadsPerBlock;

    // Llamar al kernel
    fill << <blocksPerGrid, threadsPerBlock >> > (d_v, n * m);

    hipMemcpy(posicionesVistas, d_v, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_v);
}



void create_random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup) {
    int* d_mat;
    hipMalloc(&d_mat, n * m * sizeof(int));

    hiprandState* d_state;
    hipMalloc(&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    random_matrix << <num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);
    
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_state);
}



void imprimir(int* matriz, int n, int m) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", matriz[i * m + j]);
        }
        printf("\n");
    }
}


int generarNumAleatorio(int hasta) {
    srand(generate_seed());
    int ale = rand() % (hasta + 1);
    return ale;
}


int main()
{

    int modo; //automático o manual
    int dificultad; //dificultad del juego
    int n; // número de filas
    int m; // número de columnas
    printf("Bienvenido a Cundio Crack\n");
    printf("Introduce el modo de juego con el que quieres jugar: \n 1. Automatico \n 2. Manual \n");
    scanf("%d", &modo);
    printf("Introduce la dificultad con la que quieres jugar: \n 1. Facil \n 2. Normal \n");
    scanf("%d", &dificultad);
    printf("Introduce el numero de filas que quieres que tenga el tablero: \n");
    scanf("%d", &n);
    printf("Introduce el numero de columnas que quieres que tenga el tablero: \n");
    scanf("%d", &m);

    int lim_inf = 1; // valor mínimo
    int lim_sup = 6; // valor máximo
    if (dificultad == 1) {
         lim_sup = 4; // valor máximo
    }
    
    int* mat = (int*)malloc(n * m * sizeof(int)); // matriz aleatoria
    int* posicionesVistas = (int*)malloc(n * m * sizeof(int)); 
    crear_vector(posicionesVistas, n, m);
    create_random_matrix(mat, n, m, lim_inf, lim_sup);


    //AQUI                                                                                  //<--ESTO HAY QUE VER COMO HACERLO EFICIENTE
    dim3 dimBlock(n,m);
    dim3 dimGrid(1);

    int colum=-1;
    int fila=-1;
    int dir=-1;

    while (vidas > 0) {
        imprimir(mat, n, m);
        do {
            if (modo == 2) {
                printf("Introduce la fila del caramelo que quieres comprobar\n");
                scanf("%d", &fila);
                printf("Introduce la columna del caramelo que quieres comprobar\n");
                scanf("%d", &colum);

            }
            else {                                                              //ESTA MIERDA ESTA MAL, REVISAR A VER COMO SERÍA
                srand(time(NULL));
                colum = generarNumAleatorio(n);
                srand(time(NULL));
                fila = generarNumAleatorio(m);
                printf("%d\n", colum);
                printf("%d\n", fila);
            }
        } while (colum > m && fila > n && colum < 0 && fila < 0);
        


        int elemento = mat[fila * m + colum];
        printf("Elemento antes %d \n", elemento);
        ver_candy(mat, n, m, colum, fila, posicionesVistas,elemento);
        for (int i = 0; i < n * m; ++i) {
            printf("%d ", posicionesVistas[i]);
        }
        printf("\n");

        eliminar_elementos(mat, n, m, posicionesVistas);

        vidas -= 1;
    }

   

    free(mat);

    return 0;
}