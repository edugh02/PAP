#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>


/*const int fila;
const int columna;
*/
int vidas = 5;
 //y filas, x columnas --> idy*columnas + idx ( idx es todos los .x)


// Esta función genera una semilla aleatoria basada en la hora actual.
unsigned int generate_seed() {
    time_t t;
    time(&t);
    return (unsigned int)t % 100000;
}

bool esta_en_vector(int* vector, int m, int n, int pos) {
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == pos) {
            return true;
        }
    }
    return false;
}

int primer_vacio(int* vector, int n, int m) {
    int x = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == -1) {
            x = i;
            i = n * m;
        }
    }
    return x;
}


void ver_candy(int* mat, int n, int m,int colum,int fila, int* vector,int elemento) {
    int caramelo = fila * m + colum;

    if (!esta_en_vector(vector,m,n,caramelo)&& mat[caramelo]==elemento) {
        int pos=primer_vacio(vector,n,m);
        printf("\nposicion del vector siguiente: %d \n", pos);
        vector[pos] = caramelo;
        if (fila != 0) {
            ver_candy(mat, n, m, colum, fila - 1, vector,elemento);
        }
        if (fila != n-1) {
            ver_candy(mat, n, m, colum, fila + 1, vector,elemento);
        }
        if (colum != 0) {
            ver_candy(mat, n, m, colum - 1, fila, vector,elemento);
        }
        if (colum != m-1) {
            ver_candy(mat, n, m, colum + 1, fila, vector,elemento);
        }
        
    }    
}

int cuantas_posiciones(int* vector, int n, int m) {
    int contador = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] != -1) {
            ++contador;
        }
    }
    return contador;
}

// Esta función genera una matriz aleatoria de números enteros entre "lim_inf" y "lim_sup".
__global__ void random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    // Verificar si el hilo se encuentra dentro de los límites de la matriz
    if (idx < m && idy < n) {
        // Inicializar el generador de números aleatorios
        hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
        // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
        int val = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;
        // Asignar el valor aleatorio a la matriz
        mat[idy * m + idx] = val;
    }
}

__global__ void rellenar_huecos(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
        // Calcular las coordenadas x e y del hilo
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        int idy = threadIdx.y + blockDim.y * blockIdx.y;

        // Verificar si el hilo se encuentra dentro de los límites de la matriz
        if (idx < m && idy < n && mat[idy * m + idx] == -1) {
            // Inicializar el generador de números aleatorios
            hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
            // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
            int val = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;
            
            // Asignar el valor aleatorio al hueco
            mat[idy * m + idx] = val;
            
        }
}


__global__ void eliminar_iguales_juntos(int* mat, int n, int m,int* vector) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos;

    for (int i = 0; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        } 
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}
__global__ void eliminar5(int* mat, int n, int m, int* vector,int fila,int columna) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos=fila*m+columna;
    
    if (vector[0] == idy * m + idx) {
        mat[vector[0]] = 7;
        vector[0] = -1;
    }

    for (int i = 1; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}

__global__ void eliminar6(int* mat, int n, int m, int* vector,int fila,int columna) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos = fila * m + columna;

    
    if (vector[0] == idy * m + idx) {
        mat[fila * m + columna] = 8;
        vector[0] = -1;
    }

    for (int i = 1; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}

__global__ void eliminar7oMas(int* mat, int n, int m, int* vector,int fila,int columna, unsigned int ale, hiprandState* state,int lim_sup,int lim_inf) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos;
    hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
    // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
    int val_ale = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;
    //generar numero aleatorio que indicará que tipo de r se forma
    
    if (vector[0] == idy * m + idx) {//9-->R1 10-->R2 11-->R3 21-->R4 13-->R5 14-->R6
        if (val_ale == 1) {
            mat[idy * m + idx] = 9;
        }
        else if (val_ale == 2) {
            mat[idy * m + idx] = 10;
        }
        else if (val_ale == 3) {
            mat[idy * m + idx] = 11;
        }
        else if (val_ale == 4) {
            mat[idy * m + idx] = 12;
        }
        else if (val_ale == 5) {
            mat[idy * m + idx] = 13;
        }
        else {
            mat[idy * m + idx] = 14;
        }
        vector[0] = -1;
    }

    for (int i = 1; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}

__global__  void caer_caramelos(int* matriz, int n, int m) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;


    // Contar los elementos -1 debajo del hilo
    int num_minus_1 = 0;
    for (int i = idy; i < n; ++i) {
        if (matriz[i * m + idx] == -1) {
            num_minus_1++;
        }
    }
    __syncthreads();
    // Buscar el primer elemento -1 debajo del hilo y intercambiarlo
    if (num_minus_1 > 0 && matriz[idy * m + idx] != -1) {
        int aux = matriz[idy * m + idx];
        matriz[idy * m + idx] = -1;
        matriz[(idy + num_minus_1) * m + idx] = aux;
    }
}



void caer_caramelos_host(int* matriz, int n, int m) {
    int size = n * m * sizeof(int);
    int* d_matriz;

    hipMalloc((void**)&d_matriz, size);
    hipMemcpy(d_matriz, matriz, size, hipMemcpyHostToDevice);

    // Configurar la cantidad de hilos por bloque y la cantidad de bloques por cuadrícula
    dim3 tamBloque(16, 16);
    dim3 tamCuadricula((n + tamBloque.x - 1) / tamBloque.x, (m + tamBloque.y - 1) / tamBloque.y);

    // Llamar al kernel caer_caramelos
    caer_caramelos << <tamCuadricula, tamBloque >> > (d_matriz, n, m);

    // Copiar la matriz resultante de la GPU al host
    hipMemcpy(matriz, d_matriz, size, hipMemcpyDeviceToHost);

    // Liberar memoria de la GPU
    hipFree(d_matriz);
}


__global__ void fill(int* vec, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        vec[i] = -1;
    }
}

void crear_vector(int* posicionesVistas, int n, int m) {
    int* d_v;
    hipMalloc((void**)&d_v, n * m * sizeof(int));

    // Definir la configuración del kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n * m + threadsPerBlock - 1) / threadsPerBlock;

    // Llamar al kernel
    fill << <blocksPerGrid, threadsPerBlock >> > (d_v, n * m);

    hipMemcpy(posicionesVistas, d_v, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_v);
}



void create_random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup) {
    int* d_mat;
    hipMalloc((void**)&d_mat, n * m * sizeof(int));

    hiprandState* d_state;
    hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    random_matrix << <num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);
    
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_state);
}

void rellenar_huecos_host(int* mat, int n, int m, int lim_inf, int lim_sup) {
    int* d_mat;
    hipMalloc((void**)&d_mat, n * m * sizeof(int));
    hipMemcpy(d_mat, mat, n * m * sizeof(int), hipMemcpyHostToDevice);


    hiprandState* d_state;
    hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    rellenar_huecos << <num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);

    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_state);
}


__global__ void explotarBomba(int* mat, int n, int m, int fila, int columna, unsigned int ale, hiprandState* state,int* vector) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
                                                                        //TODO falta hacer la concatenacion
    hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
    // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
    int tipo = hiprand(&state[idy * m + idx]) % 1 + 0;

    if (tipo == 0) {//Eliminar la columna entera
        if (idx == columna) {
            mat[idy * m + idx] = -1;
        }
    }
    else {//Eliminar la fila entera
        if (idy == fila) {
            mat[idy * m + idx] = -1;
        }
    }
    vector[0] = -1;
}

__global__ void explotarTNT(int* mat, int n, int m, int fila, int columna,int* vector) { 
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
                                                                            //TODO falta hacer la concatenacion
    mat[fila * m + columna] = -1;

    if (fila != 0 && idy==fila-1) {//comprobamos que no estamos en la primera fila
        if (columna != 0 && idx== columna-1) { //comprobamos que no estamos en la primera columna para borrar el elemento de la izq
            mat[idy * m + idx] = -1;
        }
        else if (columna != m && idx == columna + 1) { //comprobamos que no estamos en la ultima columna para borrar el elem de la der
            mat[idy * m + idx] = -1;
        }
        else if (idx == columna) {
            //comprobamos que que nosea la primera fila para borrar el de arriba
            if (0 < mat[idy * m + idx] && mat[idy * m + idx] < 7) {
                mat[idy * m + idx] = -1;
            }
        }
    }
    else if (fila != n && idy == fila+1) { //comprobamos que la fila no es la última
        if (columna != 0 && idx == columna-1) {//comprobamos que no estamos en la primera columna para borrar el elemento de la izq
            mat[idy * m + idx] = -1;
        }
        else if (columna != m && idx == columna + 1) { //comprobamos que no estamos en la ultima columna para borrar el elem de la der
            mat[idy * m + idx] = -1;
        }
        else if (idx == columna) {
            //comprobamos que que nosea la primera fila para borrar el de arriba
            if (0 < mat[idy * m + idx] && mat[idy * m + idx] < 7) {
                mat[idy * m + idx] = -1;
            }
        }
    }
    //para borrar el elemento de la izq, comprobamos que no estamos en la primera columna
    if (columna != 0 && idy==fila && idx == columna-1) {
        mat[idy * m + idx] = -1;
    }
    //para borrar el elemento de la der, comprobamos que no estamos en la ultima columna
    else if (columna != m && idy == fila && idx == columna + 1) {
        mat[idy * m + idx] = -1;
    }
    vector[0] = -1;
}

__global__ void explotarRx(int* matriz, int n, int m, int fila, int columna, int tipo,int *vector) { 
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int pos = idy * m + idx;
    matriz[fila*m+columna] = -1;
    switch (tipo) {
        case 9:
            if (matriz[pos] == 1) {
                matriz[pos]=-1;
            }
            break;
        case 10:
            if (matriz[pos] == 2) {
                matriz[pos] = -1;
            }
            break;
        case 11:
            if (matriz[pos] == 3) {
                matriz[pos] = -1;
            }
            break;
        case 12:
            if (matriz[pos] == 4) {
                matriz[pos] = -1;
            }
            break;
        case 13:
            if (matriz[pos] == 5) {
                matriz[pos] = -1;
            }
            break;
        case 14:
            if (matriz[pos] == 6) {
                matriz[pos] = -1;
            }
            break;
        default:
            break;
    }
    vector[0] = -1;
}

void eliminar_elementos(int* matriz, int n, int m, int* vector, int fila, int columna,int lim_sup, int lim_inf) {                       //AQUI HAY Q LLAMAR BIEN A LOS MÉTODOS DEL KERNEL
    int* d_matriz;
    int* d_vector;
    int tamVector = n * m;
    srand(time(NULL));

    // Alocar memoria para la matriz y el vector en la GPU
    hipMalloc((void**)&d_matriz, n * m * sizeof(int));
    hipMalloc((void**)&d_vector, tamVector * sizeof(int));

    // Copiar la matriz y el vector de la CPU a la GPU
    hipMemcpy(d_matriz, matriz, n * m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, vector, tamVector * sizeof(int), hipMemcpyHostToDevice);

    // Definir la configuración del kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);

    // Llamar al kernel
    switch (cuantas_posiciones(vector, n, m)) {
        //Comprobamos cuantas posiciones adyacentes hay
        case 1: 
            //No hay posciones adyacentes
            if (matriz[vector[0]] == 1 || matriz[vector[0]] == 2 || matriz[vector[0]] == 3 || matriz[vector[0]] == 4 || matriz[vector[0]] == 5 || matriz[vector[0]] == 6) {
                --vidas;
            }
            else if (matriz[vector[0]] == 7) {
                //BOMBA
                //generamos la semilla para luego crear un número aleatorio
                hiprandState* d_state;
                hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));
                unsigned int ale = generate_seed();
                explotarBomba << <gridSize, blockSize >> > (d_matriz, n, m, fila, columna, ale, d_state,d_vector);
            }
            else if (matriz[vector[0]] == 8) {
                //TNT
                explotarTNT << <gridSize, blockSize >> > (d_matriz, n, m, fila, columna,d_vector);
            }
            else if (matriz[vector[0]] > 8) {
                //Rx
                int tipo = matriz[vector[0]];
                explotarRx << <gridSize, blockSize >> > (d_matriz, n, m, fila, columna, tipo,d_vector);
            }
            vector[0] = -1;
            break;
        //Hay 1 caramelo adyacente
        case 2:
            eliminar_iguales_juntos << <gridSize, blockSize >> > (d_matriz, n, m, d_vector);
            break;
        //Hay 2 caramelos adyacentes
        case 3:
            eliminar_iguales_juntos << <gridSize, blockSize >> > (d_matriz, n, m, d_vector);
            break;
        //Hay 3 caramelos adyacentes
        case 4:
            eliminar_iguales_juntos << <gridSize, blockSize >> > (d_matriz, n, m, d_vector);
            break;
        //Hay 4 caramelos adyacentes (se boorran 5 elementos-->bomba)
        case 5:
            //Kernel sustituir el elemento de la posición por un B y borrar el resto
            //generamos la semilla para luego crear un número aleatorio
            printf("\n\nENTRA EN ELIMINAR 5\n\n");
            eliminar5 << <gridSize, blockSize >> > (d_matriz, n, m, d_vector, fila, columna);
            break;
        //Hay 5 caramelos adyacentes (se boorran 6 elementos-->TNT)
        case 6:
            //Kernel sustituir el elemento de la posición por un TNT y borrar el resto
            printf("\n\nENTRA EN ELIMINAR 6\n\n");
            eliminar6 << <gridSize, blockSize >> > (d_matriz, n, m, d_vector, fila, columna);
            break;
        //Hay 6 o mas caramelos adyacentes (se boorran 7 o mas elementos-->Rx)
        default:
            printf("\n\nENTRA EN ELIMINAR 7o+\n\n");
            //Kernel sustituir el elemento de la posición por un Rx y borrar el resto
            int tipo = rand() % 6+1;
            //generamos la semilla para luego crear un número aleatorio
            hiprandState* d_state;
            hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));
            unsigned int ale = generate_seed();
            eliminar7oMas << <gridSize, blockSize >> > (d_matriz, n, m, d_vector, fila, columna, ale, d_state,lim_sup,lim_inf);
            break;
        }


    // Copiar la matriz resultante de la GPU a la CPU
    hipMemcpy(matriz, d_matriz, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(vector, d_vector, n * m * sizeof(int), hipMemcpyDeviceToHost);

    // Liberar la memoria de la GPU
    hipFree(d_matriz);
    hipFree(d_vector);
}


void imprimir(int* matriz, int n, int m) {
    printf("\n");

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            if (matriz[i * m + j] == -1) {
                printf("    ");
            }else if (matriz[i * m + j]==7) {
                printf("B   ");
            }else if (matriz[i * m + j] == 8) {
                printf("T   ");
            }
            else if (matriz[i * m + j] == 9) {
                printf("R1  ");
            }
            else if (matriz[i * m + j] == 10) {
                printf("R2  ");
            }
            else if (matriz[i * m + j] == 11) {
                printf("R3  ");
            }
            else if (matriz[i * m + j] == 12) {
                printf("R4  ");
            }
            else if (matriz[i * m + j] == 13) {
                printf("R5  ");
            }
            else if (matriz[i * m + j] == 14) {
                printf("R6  ");
            }
            else {
                printf("%d   ", matriz[i * m + j]);
            }
        }
        printf("\n");
    }
}


int main()
{
    srand(time(NULL));
    int modo; //automático o manual
    int dificultad; //dificultad del juego
    int n; // número de filas
    int m; // número de columnas
    printf("Bienvenido a Cundio Crack\n");
    printf("Introduce el modo de juego con el que quieres jugar: \n 1. Automatico \n 2. Manual \n");
    scanf("%d", &modo);
    printf("Introduce la dificultad con la que quieres jugar: \n 1. Facil \n 2. Normal \n");
    scanf("%d", &dificultad);
    printf("Introduce el numero de filas que quieres que tenga el tablero: \n");
    scanf("%d", &n);
    printf("Introduce el numero de columnas que quieres que tenga el tablero: \n");
    scanf("%d", &m);

    int lim_inf = 1; // valor mínimo
    int lim_sup = 6; // valor máximo
    if (dificultad == 1) {
         lim_sup = 4; // valor máximo
    }
    
    int* mat = (int*)malloc(n * m * sizeof(int)); // matriz aleatoria
    int* posicionesVistas = (int*)malloc(n * m * sizeof(int)); 
    crear_vector(posicionesVistas, n, m);
    create_random_matrix(mat, n, m, lim_inf, lim_sup);

    //AQUI                                                                                  //<--ESTO HAY QUE VER COMO HACERLO EFICIENTE
    dim3 dimBlock(n,m);
    dim3 dimGrid(1);

    int colum=-1;
    int fila=-1;

    while (vidas > 0) {
        imprimir(mat, n, m);
        do {
            if (modo == 2) {
                printf("Introduce la fila del caramelo que quieres comprobar\n");
                scanf("%d", &fila);
                printf("Introduce la columna del caramelo que quieres comprobar\n");
                scanf("%d", &colum);

            }
            else {              
                colum =rand() %m;
                fila = rand() %n;
                printf("%d\n", colum);
                printf("%d\n", fila);
            }
        } while (colum > m && fila > n && colum < 0 && fila < 0);
        
        int elemento = mat[fila * m + colum];
        printf("Elemento antes %d \n", elemento);
        ver_candy(mat, n, m, colum, fila, posicionesVistas,elemento);

        for (int i = 0; i < n * m; ++i) {
            if (posicionesVistas[i] != -1) {
                printf("%d ", posicionesVistas[i]);
            }
        }

        printf("\n");

        eliminar_elementos(mat, n, m, posicionesVistas,fila,colum,lim_sup,lim_inf);

        printf("\n\n");

        imprimir(mat,n,m);

        printf("\n\n");

        caer_caramelos_host(mat, n, m);

        printf("\n\n");

        imprimir(mat, n, m);
        rellenar_huecos_host(mat, n, m, lim_inf, lim_sup);
    }

    free(mat);

    return 0;
}