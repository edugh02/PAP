#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>


/*const int fila;
const int columna;
*/
int vidas = 5;
 //y filas, x columnas --> idy*columnas + idx ( idx es todos los .x)


// Esta función genera una semilla aleatoria basada en la hora actual.
unsigned int generate_seed() {
    time_t t;
    time(&t);
    return (unsigned int)t % 100000;
}

bool esta_en_vector(int* vector, int m, int n, int pos) {
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == pos) {
            return true;
        }
    }
    return false;
}

int primer_vacio(int* vector, int n, int m) {
    int x = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == -1) {
            x = i;
            i = n * m;
        }
    }
    return x;
}


void ver_candy(int* mat, int n, int m,int colum,int fila, int* vector,int elemento) {
    int caramelo = fila * m + colum;
    //printf("\nEl caramelo esta en la posicion: %d\n", caramelo);
    //printf("Está en el vector ya? %d\n",esta_en_vector(vector, m, n, caramelo));
    //printf("El elemento es igual? %d\n", mat[caramelo] == elemento);
    //printf("Elemento dentro %d \n", mat[caramelo]);

    if ( !esta_en_vector(vector,m,n,caramelo)&& mat[caramelo]==elemento) {
        int pos=primer_vacio(vector,n,m);
        printf("\nposicion del vector siguiente: %d \n", pos);
        vector[pos] = caramelo;
        if (fila != 0) {
            ver_candy(mat, n, m, colum, fila - 1, vector,elemento);
        }
        if (fila != n) {
            ver_candy(mat, n, m, colum, fila + 1, vector,elemento);
        }
        if (colum != 0) {
            ver_candy(mat, n, m, colum - 1, fila, vector,elemento);
        }
        if (colum != m) {
            ver_candy(mat, n, m, colum + 1, fila, vector,elemento);
        }
        
    }
    
    
}

// Esta función genera una matriz aleatoria de números enteros entre "lim_inf" y "lim_sup".
__global__ void random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    // Verificar si el hilo se encuentra dentro de los límites de la matriz
    if (idx < m && idy < n) {
        // Inicializar el generador de números aleatorios
        hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
        // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
        int val = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;
        // Asignar el valor aleatorio a la matriz
        mat[idy * m + idx] = val;
    }
}



__global__ void eliminar_iguales_juntos(int* mat, int n, int m,int* vector) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos;

    for (int i = 0; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        } 
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}



void eliminar_elementos(int* matriz, int n, int m, int* vector) { //NO SE SI AQUI TMBN TENEMOS QUE ELIMINAR EL ELEMENTO DEL VECTOR DE POSICIONES 
    int* d_matriz;
    int* d_vector;
    int tamVector = n*m;

    // Alocar memoria para la matriz y el vector en la GPU
    hipMalloc(&d_matriz, n * m * sizeof(int));
    hipMalloc(&d_vector, tamVector * sizeof(int));

    // Copiar la matriz y el vector de la CPU a la GPU
    hipMemcpy(d_matriz, matriz, n * m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, vector, tamVector * sizeof(int), hipMemcpyHostToDevice);

    // Definir la configuración del kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);

    // Llamar al kernel
    eliminar_iguales_juntos << <gridSize, blockSize >> > (d_matriz, n, m, d_vector);

    // Copiar la matriz resultante de la GPU a la CPU
    hipMemcpy(matriz, d_matriz, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(vector, d_vector, n * m * sizeof(int), hipMemcpyDeviceToHost);
    
    // Liberar la memoria de la GPU
    hipFree(d_matriz);
    hipFree(d_vector);
}


__global__ void caer_caramelos(int* matriz, int n, int m) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int pos = idy * m + idx;
    int contador = 0;
    int aux;


    for (int i = idy; i < n; i++) {
        int elem_actual = i * m + idy;
        if (elem_actual == -1)contador++;
    }
    if (idy == 0) {
        aux=
        matriz[idy * m + idx + (idx * contador)] = 4;
    }
    
    
}



void caer_caramelos_host(int* matriz, int n, int m) {
    int size = n * m * sizeof(int);
    int* d_matriz;

    hipMalloc(&d_matriz, size);
    hipMemcpy(d_matriz, matriz, size, hipMemcpyHostToDevice);

    // Configurar la cantidad de hilos por bloque y la cantidad de bloques por cuadrícula
    dim3 tamBloque(16, 16);
    dim3 tamCuadricula((n + tamBloque.x - 1) / tamBloque.x, (m + tamBloque.y - 1) / tamBloque.y);

    // Llamar al kernel caer_caramelos
    caer_caramelos << <tamCuadricula, tamBloque >> > (d_matriz, n, m);

    // Copiar la matriz resultante de la GPU al host
    hipMemcpy(matriz, d_matriz, size, hipMemcpyDeviceToHost);

    // Liberar memoria de la GPU
    hipFree(d_matriz);
}


__global__ void fill(int* vec, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        vec[i] = -1;
    }
}

void crear_vector(int* posicionesVistas, int n, int m) {
    int* d_v;
    hipMalloc(&d_v, n * m * sizeof(int));

    // Definir la configuración del kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n * m + threadsPerBlock - 1) / threadsPerBlock;

    // Llamar al kernel
    fill << <blocksPerGrid, threadsPerBlock >> > (d_v, n * m);

    hipMemcpy(posicionesVistas, d_v, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_v);
}



void create_random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup) {
    int* d_mat;
    hipMalloc(&d_mat, n * m * sizeof(int));

    hiprandState* d_state;
    hipMalloc(&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    random_matrix << <num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);
    
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_state);
}



void imprimir(int* matriz, int n, int m) {
    printf("\n");

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d  ", matriz[i * m + j]);
        }
        printf("\n");
    }
}


int generarNumAleatorio(int hasta) {
    srand(generate_seed());
    int ale = rand() % (hasta + 1);
    return ale;
}


int main()
{

    int modo; //automático o manual
    int dificultad; //dificultad del juego
    int n; // número de filas
    int m; // número de columnas
    printf("Bienvenido a Cundio Crack\n");
    printf("Introduce el modo de juego con el que quieres jugar: \n 1. Automatico \n 2. Manual \n");
    scanf("%d", &modo);
    printf("Introduce la dificultad con la que quieres jugar: \n 1. Facil \n 2. Normal \n");
    scanf("%d", &dificultad);
    printf("Introduce el numero de filas que quieres que tenga el tablero: \n");
    scanf("%d", &n);
    printf("Introduce el numero de columnas que quieres que tenga el tablero: \n");
    scanf("%d", &m);

    int lim_inf = 1; // valor mínimo
    int lim_sup = 6; // valor máximo
    if (dificultad == 1) {
         lim_sup = 4; // valor máximo
    }
    
    int* mat = (int*)malloc(n * m * sizeof(int)); // matriz aleatoria
    int* posicionesVistas = (int*)malloc(n * m * sizeof(int)); 
    crear_vector(posicionesVistas, n, m);
    create_random_matrix(mat, n, m, lim_inf, lim_sup);


    //AQUI                                                                                  //<--ESTO HAY QUE VER COMO HACERLO EFICIENTE
    dim3 dimBlock(n,m);
    dim3 dimGrid(1);

    int colum=-1;
    int fila=-1;

    while (vidas > 0) {
        imprimir(mat, n, m);
        do {
            if (modo == 2) {
                printf("Introduce la fila del caramelo que quieres comprobar\n");
                scanf("%d", &fila);
                printf("Introduce la columna del caramelo que quieres comprobar\n");
                scanf("%d", &colum);

            }
            else {                                                              //ESTA MIERDA ESTA MAL, REVISAR A VER COMO SERÍA
                srand(time(NULL));
                colum = generarNumAleatorio(n);
                srand(time(NULL));
                fila = generarNumAleatorio(m);
                printf("%d\n", colum);
                printf("%d\n", fila);
            }
        } while (colum > m && fila > n && colum < 0 && fila < 0);
        


        int elemento = mat[fila * m + colum];
        printf("Elemento antes %d \n", elemento);
        ver_candy(mat, n, m, colum, fila, posicionesVistas,elemento);
        for (int i = 0; i < n * m; ++i) {
            printf("%d ", posicionesVistas[i]);
        }
        printf("\n");

        eliminar_elementos(mat, n, m, posicionesVistas);
        printf("\n\n");
        imprimir(mat,n,m);
        printf("\n\n");
        caer_caramelos_host(mat, n, m);

        vidas -= 1;
    }

   

    free(mat);

    return 0;
}