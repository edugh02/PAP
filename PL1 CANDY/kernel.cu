#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>


/*const int fila;
const int columna;
*/
int vidas = 5;
 //y filas, x columnas --> idy*columnas + idx ( idx es todos los .x)


// Esta función genera una semilla aleatoria basada en la hora actual.
unsigned int generate_seed() {
    time_t t;
    time(&t);
    return (unsigned int)t % 100000;
}

bool esta_en_vector(int* vector, int m, int n, int pos) {
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == pos) {
            return true;
        }
    }
    return false;
}

int primer_vacio(int* vector, int n, int m) {
    int x = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == -1) {
            x = i;
            i = n * m;
        }
    }
    return x;
}


void ver_candy(int* mat, int n, int m,int colum,int fila, int* vector,int elemento) {
    int caramelo = fila * m + colum;
    //printf("\nEl caramelo esta en la posicion: %d\n", caramelo);
    //printf("Está en el vector ya? %d\n",esta_en_vector(vector, m, n, caramelo));
    //printf("El elemento es igual? %d\n", mat[caramelo] == elemento);
    //printf("Elemento dentro %d \n", mat[caramelo]);

    if ( !esta_en_vector(vector,m,n,caramelo)&& mat[caramelo]==elemento) {
        int pos=primer_vacio(vector,n,m);
        printf("\nposicion del vector siguiente: %d \n", pos);
        vector[pos] = caramelo;
        if (fila != 0) {
            ver_candy(mat, n, m, colum, fila - 1, vector,elemento);
        }
        if (fila != n-1) {
            ver_candy(mat, n, m, colum, fila + 1, vector,elemento);
        }
        if (colum != 0) {
            ver_candy(mat, n, m, colum - 1, fila, vector,elemento);
        }
        if (colum != m-1) {
            ver_candy(mat, n, m, colum + 1, fila, vector,elemento);
        }
        
    }
    
    
}

int cuantas_posiciones(int* vector, int n, int m) {
    int contador = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] != -1) {
            ++contador;
        }
    }
    return contador;
}

// Esta función genera una matriz aleatoria de números enteros entre "lim_inf" y "lim_sup".
__global__ void random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    // Verificar si el hilo se encuentra dentro de los límites de la matriz
    if (idx < m && idy < n) {
        // Inicializar el generador de números aleatorios
        hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
        // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
        int val = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;
        // Asignar el valor aleatorio a la matriz
        mat[idy * m + idx] = val;
    }
}

__global__ void rellenar_huecos(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
        // Calcular las coordenadas x e y del hilo
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        int idy = threadIdx.y + blockDim.y * blockIdx.y;

        // Verificar si el hilo se encuentra dentro de los límites de la matriz
        if (idx < m && idy < n && mat[idy * m + idx] == -1) {
            // Inicializar el generador de números aleatorios
            hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
            // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
            int val = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;
            
            // Asignar el valor aleatorio al hueco
            mat[idy * m + idx] = val;
            
        }
}


__global__ void eliminar_iguales_juntos(int* mat, int n, int m,int* vector) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos;

    for (int i = 0; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        } 
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}
__global__ void eliminar5(int* mat, int n, int m, int* vector,int fila,int columna) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos=fila*m+columna;

    if (vector[0] != -1) {
        mat[fila * m + columna] = 7;
        vector[pos] = -1;
    }

    for (int i = 0; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}

void eliminar5_host(int* mat, int n, int m, int* vector, int fila, int columna) {

    int* d_mat;
    hipMalloc((void**)&d_mat, n * m * sizeof(int));
    //copiamos la matriz del host al kernel
    hipMemcpy(d_mat, mat, n * m * sizeof(int), hipMemcpyHostToDevice);

    // Definimos las dimensiones de los bloques y de la grid
    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    // Llamamos al kernel "eliminar7oMas"
    eliminar5<< <gridSize, blockSize >> > (d_mat, n, m, vector, fila, columna);

    //copiamos de vuelva la matriz del kernel al host
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
}

__global__ void eliminar6(int* mat, int n, int m, int* vector,int fila,int columna) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos = fila * m + columna;


    if (vector[0] != -1) {
        mat[fila * m + columna] = 8;
        vector[pos] = -1;
    }

    for (int i = 0; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}


void eliminar6_host(int* mat, int n, int m, int* vector, int fila, int columna) {

    int* d_mat;
    hipMalloc((void**)&d_mat, n * m * sizeof(int));
    //copiamos la matriz del host al kernel
    hipMemcpy(d_mat, mat, n * m * sizeof(int), hipMemcpyHostToDevice);

    // Definimos las dimensiones de los bloques y de la grid
    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    // Llamamos al kernel "eliminar7oMas"
    eliminar6 << <gridSize, blockSize >> > (d_mat, n, m, vector, fila, columna);

    //copiamos de vuelva la matriz del kernel al host
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
}

__global__ void eliminar7oMas(int* mat, int n, int m, int* vector,int fila,int columna, unsigned int ale, hiprandState* state) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos = fila * m + columna;

    hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
    // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
    int val_ale = hiprand(&state[idy * m + idx]) % 1 + 0;


    //generar numero aleatorio que indicará que tipo de r se forma

    if (vector[0] != -1) {  //9-->R1 10-->R2 11-->R3 21-->R4 13-->R5 14-->R6
        if (val_ale == 1) {
            mat[fila * m + columna] = 9;
        }
        else if (val_ale == 2) {
            mat[fila * m + columna] = 10;
        }
        else if (val_ale == 3) {
            mat[fila * m + columna] = 11;
        }
        else if (val_ale == 4) {
            mat[fila * m + columna] = 12;
        }
        else if (val_ale == 5) {
            mat[fila * m + columna] = 13;
        }
        else {
            mat[fila * m + columna] = 14;
        }
        vector[pos] = -1;
    }

    for (int i = 0; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}


void eliminar7oMas_host(int* mat, int n, int m, int* vector, int fila, int columna, unsigned int ale, hiprandState* state) {
    
    int* d_mat;
    hipMalloc((void**)&d_mat, n * m * sizeof(int));
    //copiamos la matriz del host al kernel
    hipMemcpy(d_mat, mat, n * m * sizeof(int), hipMemcpyHostToDevice);

    //generamos la semilla para luego crear un número aleatorio
    hiprandState* d_state;
    hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));
    unsigned int ale = generate_seed();

    // Definir las dimensiones de los bloques y de la grid
    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    // Llamar al kernel "eliminar7oMas"
    eliminar7oMas << <gridSize, blockSize >> > (d_mat, n, m, vector, fila, columna, ale, state);

    //copiamos de vuelva la matriz del kernel al host
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_state);

 }




__global__  void caer_caramelos(int* matriz, int n, int m) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;


    // Contar los elementos -1 debajo del hilo
    int num_minus_1 = 0;
    for (int i = idy; i < n; ++i) {
        if (matriz[i * m + idx] == -1) {
            num_minus_1++;
        }
    }
    __syncthreads();
    // Buscar el primer elemento -1 debajo del hilo y intercambiarlo
    if (num_minus_1 > 0 && matriz[idy * m + idx] != -1) {
        int aux = matriz[idy * m + idx];
        matriz[idy * m + idx] = -1;
        matriz[(idy + num_minus_1) * m + idx] = aux;
    }
}



void caer_caramelos_host(int* matriz, int n, int m) {
    int size = n * m * sizeof(int);
    int* d_matriz;

    hipMalloc((void**)&d_matriz, size);
    hipMemcpy(d_matriz, matriz, size, hipMemcpyHostToDevice);

    // Configurar la cantidad de hilos por bloque y la cantidad de bloques por cuadrícula
    dim3 tamBloque(16, 16);
    dim3 tamCuadricula((n + tamBloque.x - 1) / tamBloque.x, (m + tamBloque.y - 1) / tamBloque.y);

    // Llamar al kernel caer_caramelos
    caer_caramelos << <tamCuadricula, tamBloque >> > (d_matriz, n, m);

    // Copiar la matriz resultante de la GPU al host
    hipMemcpy(matriz, d_matriz, size, hipMemcpyDeviceToHost);

    // Liberar memoria de la GPU
    hipFree(d_matriz);
}


__global__ void fill(int* vec, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        vec[i] = -1;
    }
}

void crear_vector(int* posicionesVistas, int n, int m) {
    int* d_v;
    hipMalloc((void**)&d_v, n * m * sizeof(int));

    // Definir la configuración del kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n * m + threadsPerBlock - 1) / threadsPerBlock;

    // Llamar al kernel
    fill << <blocksPerGrid, threadsPerBlock >> > (d_v, n * m);

    hipMemcpy(posicionesVistas, d_v, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_v);
}



void create_random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup) {
    int* d_mat;
    hipMalloc((void**)&d_mat, n * m * sizeof(int));

    hiprandState* d_state;
    hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    random_matrix << <num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);
    
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_state);
}

void rellenar_huecos_host(int* mat, int n, int m, int lim_inf, int lim_sup) {
    int* d_mat;
    hipMalloc((void**)&d_mat, n * m * sizeof(int));
    hipMemcpy(d_mat, mat, n * m * sizeof(int), hipMemcpyHostToDevice);


    hiprandState* d_state;
    hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    rellenar_huecos << <num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);

    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_state);
}


__global__ void explotarBomba(int* mat, int n, int m, int fila, int columna, unsigned int ale, hiprandState* state) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
    // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
    int tipo = hiprand(&state[idy * m + idx]) % 1 + 0;

    if (tipo == 0) {//Eliminar la fila entera
        for (int i = 0; i < idx; ++i) {
            mat[fila * m + i] = -1;
        }
    }
    else {//Eliminar la columna entera
        for (int i = 0; i < idy; ++i) {
            mat[i * m + columna] = -1;
        }
    }
}



void explotarBomba_host(int* mat, int n, int m, int* vector, int fila, int columna) {

    int* d_mat;
    hipMalloc((void**)&d_mat, n * m * sizeof(int));
    //copiamos la matriz del host al kernel
    hipMemcpy(d_mat, mat, n * m * sizeof(int), hipMemcpyHostToDevice);

    //generamos la semilla para luego crear un número aleatorio
    hiprandState* d_state;
    hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));
    unsigned int ale = generate_seed();

    // Definimos las dimensiones de los bloques y de la grid
    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    // Llamamos al kernel "eliminar7oMas"
    explotarBomba << <gridSize, blockSize >> > (d_mat, n, m, vector, fila, columna, ale, state);

    //copiamos de vuelva la matriz del kernel al host
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
}

__global__ void explotarTNT(int* mat, int n, int m, int fila, int columna) {                                          // ESTE HAY Q CAMBIARLO Q NO LO HE QUERIDO TOCAR
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    mat[fila * m + columna] = -1;
    if (fila != 0) {
        if (columna != 0) {
            mat[(fila - 1) * m + columna - 1] = -1;
        }
        else if (columna != m - 1) {
            mat[(fila - 1) * m + columna + 1] = -1; 
        }
        if (0 < mat[(fila - 1) * m + columna] && mat[(fila - 1) * m + columna] < 7) {
            mat[(fila - 1) * m + columna] = -1;
        }
        else if (mat[(fila - 1) * m + columna] == 7) {
            explotarBomba(mat, n, m, fila - 1, columna);
        }
        else if (mat[(fila - 1) * m + columna] == 8) {
            explotarTNT(mat, n, m, fila - 1, columna);
        }
        else if (mat[(fila - 1) * m + columna] > 8) {
            explotarRx(mat, n, m, fila - 1, columna, mat[(fila - 1) * m + columna]);
        }
    }
    else if (fila != n - 1) {
        if (columna != 0) {
            mat[(fila + 1) * m + columna - 1] = -1;
        }
        else if (columna != m - 1) {
            mat[(fila + 1) * m + columna + 1] = -1;
        }
        mat[(fila + 1) * m + columna] = -1;
    }
    if (columna != 0) {
        mat[fila * m + columna - 1] = -1;
    }
    else if (columna != m - 1) {
        mat[fila * m + columna + 1] = -1;
    }
}


void explotarTNT_host(int* mat, int n, int m, int* vector, int fila, int columna) {

    int* d_mat;
    hipMalloc((void**)&d_mat, n * m * sizeof(int));
    //copiamos la matriz del host al kernel
    hipMemcpy(d_mat, mat, n * m * sizeof(int), hipMemcpyHostToDevice);

    // Definimos las dimensiones de los bloques y de la grid
    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    // Llamamos al kernel "eliminar7oMas"
    explotarTNT << <gridSize, blockSize >> > (d_mat, n, m, vector, fila, columna);

    //copiamos de vuelva la matriz del kernel al host
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
}

__global__ void explotarRx(int* matriz, int n, int m, int fila, int columna, int tipo) {                   //HACER HOST PARA ESTA, NO SABIA COMO SACAR EL TIPO
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int pos = idy * m + idy;
    switch (tipo) {
    case 9:
        if (matriz[pos] == 1) {
            matriz[pos = -1];
        }
        break;
    case 10:
        if (matriz[pos] == 2) {
            matriz[pos = -1];
        }
        break;
    case 11:
        if (matriz[pos] == 3) {
            matriz[pos = -1];
        }
        break;
    case 12:
        if (matriz[pos] == 4) {
            matriz[pos = -1];
        }
        break;
    case 13:
        if (matriz[pos] == 5) {
            matriz[pos = -1];
        }
        break;
    case 14:
        if (matriz[pos] == 6) {
            matriz[pos = -1];
        }
        break;
    default:
        break;
    }
}



void eliminar_elementos(int* matriz, int n, int m, int* vector, int fila, int columna) {                       //AQUI HAY Q LLAMAR BIEN A LOS MÉTODOS DEL KERNEL
    int* d_matriz;
    int* d_vector;
    int tamVector = n * m;

    // Alocar memoria para la matriz y el vector en la GPU
    hipMalloc((void**)&d_matriz, n * m * sizeof(int));
    hipMalloc((void**)&d_vector, tamVector * sizeof(int));

    // Copiar la matriz y el vector de la CPU a la GPU
    hipMemcpy(d_matriz, matriz, n * m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, vector, tamVector * sizeof(int), hipMemcpyHostToDevice);

    // Definir la configuración del kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);

    // Llamar al kernel
    switch (cuantas_posiciones(vector, n, m)) {
    case 1: //TENEMOS QUE ACORDARNOS DE BORRAR LA PRIMERA POSICION DEL VECTOR, QUE SE HABRÁ RELLENADO
        if (matriz[vector[0]] == 1 || matriz[vector[0]] == 2 || matriz[vector[0]] == 3 || matriz[vector[0]] == 4 || matriz[vector[0]] == 5 || matriz[vector[0]] == 6) {
            --vidas;
        }
        else if (matriz[vector[0]] == 7) {
            //BOMBA
            explotarBomba(matriz, n, m, fila, columna);
        }
        else if (matriz[vector[0]] == 8) {
            //TNT
            explotarTNT(matriz, n, m, fila, columna);
        }
        else if (matriz[vector[0]] > 8) {
            //Rx
            explotarRx(matriz, n, m, fila, columna, matriz[fila * m + columna]);
        }
        vector[0] = -1;
        break;
    case 2:
        eliminar_iguales_juntos << <gridSize, blockSize >> > (d_matriz, n, m, d_vector);
        break;
    case 3:
        eliminar_iguales_juntos << <gridSize, blockSize >> > (d_matriz, n, m, d_vector);
        break;
    case 4:
        eliminar_iguales_juntos << <gridSize, blockSize >> > (d_matriz, n, m, d_vector);
        break;
    case 5:
        //Kernel sustituir el elemento de la posición por un B y borrar el resto
        eliminar5 << <gridSize, blockSize >> > (d_matriz, n, m, d_vector, fila, columna);
        break;
    case 6:
        //Kernel sustituir el elemento de la posición por un TNT y borrar el resto
        eliminar6 << <gridSize, blockSize >> > (d_matriz, n, m, d_vector, fila, columna);
        break;
    default:
        //Kernel sustituir el elemento de la posición por un Rx y borrar el resto
        eliminar7oMas << <gridSize, blockSize >> > (d_matriz, n, m, d_vector, fila, columna);
        break;
    }


    // Copiar la matriz resultante de la GPU a la CPU
    hipMemcpy(matriz, d_matriz, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(vector, d_vector, n * m * sizeof(int), hipMemcpyDeviceToHost);

    // Liberar la memoria de la GPU
    hipFree(d_matriz);
    hipFree(d_vector);
}


void imprimir(int* matriz, int n, int m) {
    printf("\n");

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            if (matriz[i * m + j]==7) {
                printf("B   ");
            }else if (matriz[i * m + j] == 8) {
                printf("T   ");
            }
            else if (matriz[i * m + j] == 9) {
                printf("R1  ");
            }
            else if (matriz[i * m + j] == 10) {
                printf("R2  ");
            }
            else if (matriz[i * m + j] == 11) {
                printf("R3  ");
            }
            else if (matriz[i * m + j] == 12) {
                printf("R4  ");
            }
            else if (matriz[i * m + j] == 13) {
                printf("R5  ");
            }
            else if (matriz[i * m + j] == 14) {
                printf("R6  ");
            }
            else {
                printf("%d   ", matriz[i * m + j]);
            }
        }
        printf("\n");
    }
}


int main()
{
    srand(time(NULL));
    int modo; //automático o manual
    int dificultad; //dificultad del juego
    int n; // número de filas
    int m; // número de columnas
    printf("Bienvenido a Cundio Crack\n");
    printf("Introduce el modo de juego con el que quieres jugar: \n 1. Automatico \n 2. Manual \n");
    scanf("%d", &modo);
    printf("Introduce la dificultad con la que quieres jugar: \n 1. Facil \n 2. Normal \n");
    scanf("%d", &dificultad);
    printf("Introduce el numero de filas que quieres que tenga el tablero: \n");
    scanf("%d", &n);
    printf("Introduce el numero de columnas que quieres que tenga el tablero: \n");
    scanf("%d", &m);

    int lim_inf = 1; // valor mínimo
    int lim_sup = 6; // valor máximo
    if (dificultad == 1) {
         lim_sup = 2; // valor máximo
    }
    
    int* mat = (int*)malloc(n * m * sizeof(int)); // matriz aleatoria
    int* posicionesVistas = (int*)malloc(n * m * sizeof(int)); 
    crear_vector(posicionesVistas, n, m);
    create_random_matrix(mat, n, m, lim_inf, lim_sup);


    //AQUI                                                                                  //<--ESTO HAY QUE VER COMO HACERLO EFICIENTE
    dim3 dimBlock(n,m);
    dim3 dimGrid(1);

    int colum=-1;
    int fila=-1;

    while (vidas > 0) {
        imprimir(mat, n, m);
        do {
            if (modo == 2) {
                printf("Introduce la fila del caramelo que quieres comprobar\n");
                scanf("%d", &fila);
                printf("Introduce la columna del caramelo que quieres comprobar\n");
                scanf("%d", &colum);

            }
            else {              
                colum =rand() %m;
                fila = rand() %n;
                printf("%d\n", colum);
                printf("%d\n", fila);
            }
        } while (colum > m && fila > n && colum < 0 && fila < 0);
        


        int elemento = mat[fila * m + colum];
        printf("Elemento antes %d \n", elemento);
        ver_candy(mat, n, m, colum, fila, posicionesVistas,elemento);
        for (int i = 0; i < n * m; ++i) {
            printf("%d ", posicionesVistas[i]);
        }
        printf("\n");

        eliminar_elementos(mat, n, m, posicionesVistas,fila,colum);
        printf("\n\n");
        imprimir(mat,n,m);
        printf("\n\n");
        caer_caramelos_host(mat, n, m);
        imprimir(mat, n, m);
        rellenar_huecos_host(mat, n, m, lim_inf, lim_sup);
    }

    free(mat);

    return 0;
}