#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>


/*const int fila;
const int columna;
*/
int vidas=5;



unsigned int generate_seed() {
    time_t t;
    time(&t);
    return (unsigned int)t%100000;
}


__global__ void random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup,unsigned int ale, hiprandState* state) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx < n && idy < m) {
        hiprand_init(ale, idx * m + idy, 0, &state[idx * m + idy]); // inicializar el generador de números aleatorios
        int val = hiprand(&state[idx * m + idy]) % lim_sup + lim_inf; // generar un número aleatorio entero entre 1 y 6
        mat[idx * m + idy] = val;
    }
}

void create_random_matrix(int* mat, int n, int m, int lim_inf, int lim_sup) {
    int* d_mat;
    hipMalloc(&d_mat, n * m * sizeof(int));

    hiprandState* d_state;
    hipMalloc(&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    dim3 block_size(32, 32);
    dim3 num_blocks((n + block_size.x - 1) / block_size.x, (m + block_size.y - 1) / block_size.y);
    random_matrix << <num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);

    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_mat);
    hipFree(d_state);
}



void imprimir(int* matriz, int n, int m) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", matriz[i * m + j]);
        }
        printf("\n");
    }
}

int main()
{   

    /*
    int N = 10;
    int *d_a,*d_b,*d_c,*d_d;
    int matriz_host[N][N];

    printf("Bienvenido a Cundio Crack\n");
    printf("Introduce el modo de juego con el que quieres jugar: \n 1. Automático \n 2. Manual \n");
    int m = scanf("%d", &m);
    printf("Introduce la dificultad con la que quieres jugar: \n 1. Facil \n 2. Normal \n");
    int d = scanf("%d", &d);
    printf("Introduce el numero de filas que quieres que tenga el tablero: \n");
    int fi = scanf("%d", &fi);
    printf("Introduce el numero de columnas que quieres que tenga el tablero: \n");
    int co = scanf("%d", &co);
*/




    // método para crear una matriz aleatoria de n * m 
    int n = 10; // número de filas
    int m = 5; // número de columnas
    int lim_inf = 1; // valor mínimo
    int lim_sup = 6; // valor máximo
    int* mat = (int*)malloc(n * m * sizeof(int)); // matriz aleatoria

    create_random_matrix(mat, n, m, lim_inf, lim_sup);

    imprimir(mat,n,m);

    // hacer algo con la matriz aleatoria

    free(mat);



    return 0;
}