///////////////////////////////////////////////////////////////////////////
// includes
///////////////////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>



///////////////////////////////////////////////////////////////////////////
// defines
const int N = 20;
const int TAM_MASK = 7;
const int R = TAM_MASK/2;

const int SIZE = N * sizeof(int);
const int SIZE_MASK = TAM_MASK * sizeof(int);

__constant__ int mascara[TAM_MASK];

///////////////////////////////////////////////////////////////////////////
// declaracion de funciones

// DEVICE: funcion llamada desde el device y ejecutada en el device

// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void convulcion1D(int* dev_A, int* dev_R){
	int pos = threadIdx.x;

	int acum = 0;
    int mask = 0;
    int inicio, fin;

    if (pos < R)
    {
        mask = R - pos;
        inicio = 0;
        fin = pos + 2;
    }
    else if (pos >= N - 2)
    {
        inicio = pos - 2;
        fin = N - 1;
    }
    else
    {
        inicio = pos - 2;
        fin = pos + 2;
    }
    for (int i = inicio; i <= fin; i++)
    {
        acum += dev_A[i] * mascara[mask];
        mask++;
    }
    dev_R[pos] = acum;
}

// HOST: funcion llamada desde el host y ejecutada en el host

///////////////////////////////////////////////////////////////////////////
// MAIN: rutina principal ejecutada en el host
int main() 
{
	// cuerpo del programa
	printf(" << CONVOLUCION 1D >>");

	//Instanciamos los vectores y le llenamos de valores manualmente
	int h_A[N];
	printf("\n A \n");
	for (int x = 0; x < N; x++){
		h_A[x] = x + 1;
		printf("%d ", h_A[x]);
	}
	printf("\n");

	int h_M[TAM_MASK] = {3, 4, 5, 6, 5, 4, 3};
	int h_R[N];

	//Puntero que usar� la GPU 
	int* dev_A, * dev_R;

	//Reservamos memoria para GPU
	hipMalloc((void**)&dev_A, SIZE);
	hipMalloc((void**)&dev_R, SIZE);

	hipMemcpyToSymbol(HIP_SYMBOL(mascara),&h_M, SIZE_MASK);

	//Copiamos a la GPU
	hipMemcpy(dev_A, h_A, SIZE, hipMemcpyHostToDevice);

	//POR BLOQUES	
	dim3 threadsInBlock(N);
	dim3 blocksInGrid(1);
	printf("\nBlocksInGrid -> %dx%dx%d\nThreadsInBlock -> %dx%dx%d", 
		blocksInGrid.x, blocksInGrid.y, blocksInGrid.z, threadsInBlock.x, threadsInBlock.y, threadsInBlock.z);
	convulcion1D <<<blocksInGrid, threadsInBlock >>> (dev_A, dev_R);

	//Recuperamos el resultado de la GPU
	hipMemcpy(h_R, dev_R, SIZE , hipMemcpyDeviceToHost);

	printf("\n\n -- MASCARA -- \n");
    for (int i = 0; i < TAM_MASK; i++)
    {
        printf("%d ", h_M[i]);
    }

	printf("\n\n -- Resultado -- \n");
	for (int x = 0; x < N; x++){
		printf("%d ", h_R[x]);
	}
	printf("\n");

	hipFree(dev_A); 
	hipFree(dev_R);

	// salida del programa
	printf("\npulsa INTRO para finalizar...");
	fflush(stdin);
	char tecla = getchar();
	return 0;
}
///////////////////////////////////////////////////////////////////////////