#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define SH_DIM_Y 16
#define SH_DIM_X 16
int vidas = 5;
 //y filas, x columnas --> idy*columnas + idx ( idx es todos los .x)

//inicialización tonta para ver que luego cambia
int HILOS_BLOQUE_X=777;
int HILOS_BLOQUE_Y=777;
int BLOQUES_GRID_X=777;
int BLOQUES_GRID_Y=777;


// Esta función genera una semilla aleatoria basada en la hora actual.
unsigned int generate_seed() {
    time_t t;
    time(&t);
    return (unsigned int)t % 100000;
}

//Comprueba si una posicion esta en el vector
bool esta_en_vector(int* vector, int m, int n, int pos) {
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == pos) {
            return true;
        }
    }
    return false;
}

//retorna la primera poscion donde haya un -1
int primer_vacio(int* vector, int n, int m) {
    int x = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == -1) {
            x = i;
            i = n * m;
        }
    }
    return x;
}


void ver_candy(int* mat, int n, int m,int colum,int fila, int* vector,int elemento) {
    int caramelo = fila * m + colum;//posicion en la matriz de las coordenadas

    if (!esta_en_vector(vector,m,n,caramelo)&& mat[caramelo]==elemento) {
        //comprobamos que la posicion no ha sido ya insertada
        int pos=primer_vacio(vector,n,m);
        vector[pos] = caramelo;
        //insertamos en la primera posicion que se encuentre vacia del vector, la posicion del caramelo
        if (fila != 0) {//Adyacente de arriba
            ver_candy(mat, n, m, colum, fila - 1, vector,elemento);
        }
        if (fila != n - 1) {//Adyacente de abajo
            ver_candy(mat, n, m, colum, fila + 1, vector,elemento);
        }
        if (colum != 0) {//Adyacente de la izquierda
            ver_candy(mat, n, m, colum - 1, fila, vector,elemento);
        }
        if (colum != m - 1) {//Adyacente de la derecha
            ver_candy(mat, n, m, colum + 1, fila, vector,elemento);
        }
        
    }    
}

//obtencion de cuantas posiciones han sido encontradas para ser borradas
int cuantas_posiciones(int* vector, int n, int m) {
    int contador = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] != -1) {
            ++contador;
        }
    }
    return contador;
}

// Esta función genera una matriz aleatoria de números enteros entre "lim_inf" y "lim_sup".
__global__ void matriz_aleatoria(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    // Verificar si el hilo se encuentra dentro de los límites de la matriz
    if (idx < m && idy < n) {
        // Inicializar el generador de números aleatorios
        hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
        // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
        int val = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;
        // Asignar el valor aleatorio a la matriz
        mat[idy * m + idx] = val;
    }
}

__global__ void rellenar_huecos(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    // Calcular el índice global del hilo
    int id = idy * m + idx;

    // Calcular el índice local del hilo dentro del bloque
    int loc_id = threadIdx.y * blockDim.x + threadIdx.x;

    // Declarar la matriz compartida
    __shared__ int sh_mat[SH_DIM_Y * SH_DIM_X];

    // Inicializar la matriz compartida
    if (loc_id < SH_DIM_X * SH_DIM_Y) {
        int sh_id = (threadIdx.y + 1) * SH_DIM_X + threadIdx.x + 1;
        if (idx >= m || idy >= n) {
            sh_mat[sh_id] = 0;
        }
        else {
            sh_mat[sh_id] = mat[id];
        }
    }

    // Sincronizar los hilos dentro del bloque para asegurar que la matriz compartida se inicializa correctamente
    __syncthreads();

    // Verificar si el hilo se encuentra dentro de los límites de la matriz
    if (idx < m && idy < n && mat[id] == -1) {
        // Inicializar el generador de números aleatorios
        hiprand_init(ale, id, 0, &state[id]);

        // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
        int val = hiprand(&state[id]) % lim_sup + lim_inf;

        // Asignar el valor aleatorio al hueco
        sh_mat[(threadIdx.y + 1) * SH_DIM_X + threadIdx.x + 1] = val;

        // Sincronizar los hilos dentro del bloque para asegurar que todos los valores aleatorios se han generado antes de escribirlos en la matriz global
        __syncthreads();

        // Asignar los valores aleatorios generados a la matriz global
        mat[id] = sh_mat[(threadIdx.y + 1) * SH_DIM_X + threadIdx.x + 1];
    }
}


__global__ void eliminar_iguales_juntos(int* mat, int n, int m, int* vector) {
    // Definir la memoria compartida
    __shared__ int s_vector[SH_DIM_X * SH_DIM_Y];

    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;

    // Copiar el vector a la memoria compartida
    if (tid < n * m) {
        s_vector[tid] = vector[tid];
    }
    __syncthreads();

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (idx < m && idy < n) {
        for (int i = 0; i < n * m; i++) {
            if (s_vector[i] == idy * m + idx) {
                mat[idy * m + idx] = -1;
                vector[i] = -1;
                break;
            }
        }
    }
}


__global__ void eliminar5(int* mat, int n, int m, int* vector,int fila,int columna) {
    // Definir la memoria compartida
    __shared__ int s_vector[SH_DIM_X * SH_DIM_Y];

    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;

    bool centinela = false;
    int pos = fila * m + columna;
    // Copiar el vector a la memoria compartida
    if (tid < n * m) {
        s_vector[tid] = vector[tid];
    }
    __syncthreads();

    if (s_vector[0] == idy * m + idx) {
        mat[vector[0]] = 7;//Se pone una bomba en la posicion
        vector[0] = -1;
    }

    for (int i = 1; i < n * m; i++) {
        if (s_vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}

__global__ void eliminar6(int* mat, int n, int m, int* vector, int fila, int columna) {
    // Definir la memoria compartida
    __shared__ int s_vector[SH_DIM_X * SH_DIM_Y];

    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;

    bool centinela = false;
    int pos = fila * m + columna;

    // Copiar el vector a la memoria compartida
    if (tid < n * m) {
        s_vector[tid] = vector[tid];
    }
    __syncthreads();

    if (s_vector[0] == idy * m + idx) {
        mat[fila * m + columna] = 8;//se pone una TNT en la posicion 
        s_vector[0] = -1;
    }

    for (int i = 1; i < n * m; i++) {
        if (s_vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        s_vector[pos] = -1;
    }

    // Copiar el vector de vuelta a la memoria global
    if (tid < n * m) {
        vector[tid] = s_vector[tid];
    }
}

__global__ void eliminar7oMas(int* mat, int n, int m, int* vector, int fila, int columna, unsigned int ale, hiprandState* state, int lim_sup, int lim_inf) {
    // Declarar la memoria compartida
    extern __shared__ int shared_mem[];
    int* mat_shared = shared_mem;
    int* vector_shared = &mat_shared[n * m];

    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;

    // Copiar los datos a la memoria compartida teniendo en cuenta que no estén fuera de rango
    if (idx >= 0 && idx < m && idy >= 0 && idy < n) {
        mat_shared[idy * m + idx] = mat[idy * m + idx];
    }

    if (tid < n * m) {
        vector_shared[tid] = vector[tid];
    }
    
    __syncthreads();

    hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);

    // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
    int val_ale = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;

    //generar numero aleatorio que indicará que tipo de r se forma
    if (vector_shared[0] == idy * m + idx) {//9-->R1 10-->R2 11-->R3 21-->R4 13-->R5 14-->R6
        if (val_ale == 1) {//En la posicion se pone un R1
            mat_shared[idy * m + idx] = 9;
        }
        else if (val_ale == 2) {//En la posicion se pone un R2
            mat_shared[idy * m + idx] = 10;
        }
        else if (val_ale == 3) {//En la posicion se pone un R3
            mat_shared[idy * m + idx] = 11;
        }
        else if (val_ale == 4) {//En la posicion se pone un R4
            mat_shared[idy * m + idx] = 12;
        }
        else if (val_ale == 5) {//En la posicion se pone un R5
            mat_shared[idy * m + idx] = 13;
        }
        else {//En la posicion se pone un R6
            mat_shared[idy * m + idx] = 14;
        }
        vector_shared[0] = -1;
    }

    __syncthreads();

    for (int i = 1; i < n * m; i++) {
        if (vector_shared[i] == idy * m + idx) {// El número está presente en el vector
            mat_shared[idy * m + idx] = -1;
            vector_shared[i] = -1;
        }
    }

    // Copiar los datos actualizados de vuelta a la memoria global
    if (idx >= 0 && idx < m && idy >= 0 && idy < n) {
        mat[idy * m + idx] = mat_shared[idy * m + idx];
    }
    if (tid < n * m) {
        vector[idx] = vector_shared[idx];
    }
    
    __syncthreads();
}


__global__ void caer_caramelos(int* matriz, int n, int m) {
    // Declarar la memoria compartida
    extern __shared__ int shared_mem[];
    int* matriz_shared = shared_mem;

    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    // Copiar los datos a la memoria compartida
    matriz_shared[idy * m + idx] = matriz[idy * m + idx];
    __syncthreads();

    // Contar los elementos -1 debajo del hilo
    int num_minus_1 = 0;
    for (int i = idy; i < n; ++i) {
        if (matriz_shared[i * m + idx] == -1) {
            num_minus_1++;
        }
    }

    __syncthreads();
    // Buscar el primer elemento -1 debajo del hilo y intercambiarlo
    if (num_minus_1 > 0 && matriz_shared[idy * m + idx] != -1) {
        int aux = matriz_shared[idy * m + idx];
        matriz_shared[idy * m + idx] = -1;
        matriz_shared[(idy + num_minus_1) * m + idx] = aux;
    }

    __syncthreads();

    // Copiar los datos actualizados de vuelta a la memoria global
    matriz[idy * m + idx] = matriz_shared[idy * m + idx];
}




void caer_caramelos_host(int* matriz, int n, int m) {
    //Creacion de puntero para la GPU
    int* d_matriz;

    //Reservamos memoria 
    int size = n * m * sizeof(int);
    hipMalloc((void**)&d_matriz, size);

    //copiamos del host al device
    hipMemcpy(d_matriz, matriz, size, hipMemcpyHostToDevice);

    // Configurar la cantidad de hilos por bloque y la cantidad de bloques por cuadrícula
    dim3 block_size(HILOS_BLOQUE_X, HILOS_BLOQUE_Y);
    dim3 num_blocks(BLOQUES_GRID_X, BLOQUES_GRID_Y);

    // Llamar al kernel caer_caramelos
    caer_caramelos <<<num_blocks, block_size >> > (d_matriz, n, m);

    // Copiar la matriz resultante de la GPU al host
    hipMemcpy(matriz, d_matriz, size, hipMemcpyDeviceToHost);

    // Liberar memoria de la GPU
    hipFree(d_matriz);
}


__global__ void rellenar(int* vec, int n,int m) {
    //rellenar el vector con -1

    //posicion del hilo correspondiente al vector
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    if (idy < n && idx < m) {
        vec[idy*m+idx] = -1;
    }
}

void crear_vector(int* posicionesVistas, int n, int m) {
    //creacion de un puntero para la GPU
    int* d_v;
    //Reservamos memoria
    hipMalloc((void**)&d_v, n * m * sizeof(int));

    // Definir la configuración del kernel 
    dim3 block_size(HILOS_BLOQUE_X, HILOS_BLOQUE_Y);
    dim3 num_blocks(BLOQUES_GRID_X, BLOQUES_GRID_Y);

    // Llamar al kernel
    rellenar <<<num_blocks, block_size >> > (d_v, n, m);

    //Copiamos en el host el vector creado por el kernel
    hipMemcpy(posicionesVistas, d_v, n * m * sizeof(int), hipMemcpyDeviceToHost);
    //liberamos el puntero
    hipFree(d_v);
}



void crear_matriz_aleatoria(int* mat, int n, int m, int lim_inf, int lim_sup) {
    //creacion de un puntero para la GPU
    int* d_mat;
    //reservamos espacio en la memoria
    hipMalloc((void**)&d_mat, n * m * sizeof(int));

    //puntero que ayudara a la obtencion de numeros aleatorios dentro del Kernel
    hiprandState* d_state;
    //reservamos espacio en la memoria
    hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));

    //obtencion de una semilla que ayudara a la creacion de numeros aleatorios
    unsigned int ale = generate_seed();

    dim3 block_size(HILOS_BLOQUE_X, HILOS_BLOQUE_Y);
    dim3 num_blocks(BLOQUES_GRID_X, BLOQUES_GRID_Y);
    matriz_aleatoria <<<num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);
    
    //Copiamos en el host el resultado obtenido en el device
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    //liberacion de punteros
    hipFree(d_mat);
    hipFree(d_state);
}

void rellenar_huecos_host(int* mat, int n, int m, int lim_inf, int lim_sup) {
    //creacion de un puntero para la GPU
    int* d_mat;

    //Reservamos memoria 
    hipMalloc((void**)&d_mat, n * m * sizeof(int));

    //Copiamos del host al device 
    hipMemcpy(d_mat, mat, n * m * sizeof(int), hipMemcpyHostToDevice);

    //Creacion de un puntero que ayudara a la creacion de numeros aleatorios
    hiprandState* d_state;
    hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    dim3 block_size(HILOS_BLOQUE_X, HILOS_BLOQUE_Y);
    dim3 num_blocks(BLOQUES_GRID_X, BLOQUES_GRID_Y);
    rellenar_huecos <<<num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);

    //Copiamos del device al host
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    //liberamos punteros
    hipFree(d_mat);
    hipFree(d_state);
}


__global__ void explotarBomba(int* mat, int n, int m, int fila, int columna, int tipo, int* vector) {
    int idx = threadIdx.x;
    int idy = threadIdx.y;

    if (tipo == 0) {//Eliminar la columna entera
        if (idx == columna) {
            mat[idy * m + idx] = -1;
        }
    }
    else {//Eliminar la fila entera
        if (idy == fila) {
            mat[idy * m + idx] = -1;
        }
    }
    //el elemento deja de ser analizado
    vector[0] = -1;
}


__global__ void explotarTNT(int* mat, int n, int m, int fila, int columna, int* vector) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;//coordenada de la fila del hilo
    int idy = threadIdx.y + blockDim.y * blockIdx.y;//coordenada de la columna del hilo

    //Comprobacion de que la posicion del hilo esta dentro de la matriz
    if (idx >= 0 && idx < m && idy >= 0 && idy < n) {
        //Comprobacion de que la posicion del hilo este dentro del radio de explosion
        if ((idx >= columna - 4 && idx <= columna + 4) && (idy >= fila - 4 && idy <= fila + 4)) {
            mat[idy * m + idx] = -1;
        }
    }

    //el elemento deja de ser analizado
    vector[0] = -1;
}

__global__ void explotarRx(int* matriz, int n, int m, int fila, int columna, int tipo,int *vector) { 
    int idx = threadIdx.x + blockDim.x * blockIdx.x;//coordenada de la fila del hilo
    int idy = threadIdx.y + blockDim.y * blockIdx.y;//coordenada de la columna del hilo
    int pos = idy * m + idx;//posicion del hilo
    matriz[fila*m+columna] = -1;
    switch (tipo) {
        case 9://R1-->eliminacion de todos los 1 de la matriz
            if (matriz[pos] == 1) {
                matriz[pos]=-1;
            }
            break;
        case 10://R2-->eliminacion de todos los 2 de la matriz
            if (matriz[pos] == 2) {
                matriz[pos] = -1;
            }
            break;
        case 11://R3-->eliminacion de todos los 3 de la matriz
            if (matriz[pos] == 3) {
                matriz[pos] = -1;
            }
            break;
        case 12://R4-->eliminacion de todos los 4 de la matriz
            if (matriz[pos] == 4) {
                matriz[pos] = -1;
            }
            break;
        case 13://R5-->eliminacion de todos los 5 de la matriz
            if (matriz[pos] == 5) {
                matriz[pos] = -1;
            }
            break;
        case 14://R6-->eliminacion de todos los 6 de la matriz
            if (matriz[pos] == 6) {
                matriz[pos] = -1;
            }
            break;
        default:
            break;
    }
    //el elemento deja de ser analizado
    vector[0] = -1;
}

__global__ void eliminar1(int* mat, int* vector, int n, int m, int fila, int columna) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;//coordenada de la fila del hilo
    int idy = threadIdx.y + blockDim.y * blockIdx.y;//coordenada de la columna del hilo
    int pos = idy * m + idx;//posicion del hilo
    if (pos == fila * m + columna) {
        vector[0] = -1;
        mat[pos] = -1;
    }
}

void eliminar_elementos(int* matriz, int n, int m, int* vector, int fila, int columna,int lim_sup, int lim_inf) {                       //AQUI HAY Q LLAMAR BIEN A LOS MÉTODOS DEL KERNEL
    //creacion de punteros para la GPU
    int* d_matriz;
    int* d_vector;

    int tamVector = n * m;
    srand(time(NULL));

    // Alocar memoria para la matriz y el vector en la GPU
    hipMalloc((void**)&d_matriz, n * m * sizeof(int));
    hipMalloc((void**)&d_vector, tamVector * sizeof(int));

    // Copiar la matriz y el vector de la CPU a la GPU
    hipMemcpy(d_matriz, matriz, n * m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, vector, tamVector * sizeof(int), hipMemcpyHostToDevice);

    // Definir la configuración del kernel
    dim3 block_size(HILOS_BLOQUE_X, HILOS_BLOQUE_Y);
    dim3 num_blocks(BLOQUES_GRID_X, BLOQUES_GRID_Y);

    // Llamar al kernel
    switch (cuantas_posiciones(vector, n, m)) {
        //Comprobamos cuantas posiciones adyacentes hay
        case 1: 
            //No hay posciones adyacentes
            if (matriz[vector[0]] == 1 || matriz[vector[0]] == 2 || matriz[vector[0]] == 3 || matriz[vector[0]] == 4 || matriz[vector[0]] == 5 || matriz[vector[0]] == 6) {
                vidas--;
                eliminar1 << <num_blocks, block_size >> > (d_matriz, d_vector, n, m, fila, columna);
            }
            else if (matriz[vector[0]] == 7) {
                //BOMBA
                //generamos la semilla para luego crear un número aleatorio
                int ale = rand() % 2;
                explotarBomba <<<num_blocks, block_size >> > (d_matriz, n, m, fila, columna, ale,d_vector);
            }
            else if (matriz[vector[0]] == 8) {
                //TNT
                explotarTNT <<<num_blocks, block_size >> > (d_matriz, n, m, fila, columna,d_vector);
            }
            else if (matriz[vector[0]] > 8) {
                //Rx
                int tipo = matriz[vector[0]];
                explotarRx <<<num_blocks, block_size >> > (d_matriz, n, m, fila, columna, tipo,d_vector);
            }
            break;
        //Hay 1 caramelo adyacente
        case 2:
            eliminar_iguales_juntos <<<num_blocks, block_size >> > (d_matriz, n, m, d_vector);
            break;
        //Hay 2 caramelos adyacentes
        case 3:
            eliminar_iguales_juntos <<<num_blocks, block_size >> > (d_matriz, n, m, d_vector);
            break;
        //Hay 3 caramelos adyacentes
        case 4:
            eliminar_iguales_juntos <<<num_blocks, block_size >> > (d_matriz, n, m, d_vector);
            break;
        //Hay 4 caramelos adyacentes (se boorran 5 elementos-->bomba)
        case 5:
            //Kernel sustituir el elemento de la posición por un B y borrar el resto
            eliminar5 <<<num_blocks, block_size >> > (d_matriz, n, m, d_vector, fila, columna);
            break;
        //Hay 5 caramelos adyacentes (se boorran 6 elementos-->TNT)
        case 6:
            //Kernel sustituir el elemento de la posición por un TNT y borrar el resto
            eliminar6 <<<num_blocks, block_size >> > (d_matriz, n, m, d_vector, fila, columna);
            break;
        //Hay 6 o mas caramelos adyacentes (se boorran 7 o mas elementos-->Rx)
        default:
            //Kernel sustituir el elemento de la posición por un Rx y borrar el resto
            int tipo = rand() % lim_sup+lim_inf;
            //generamos la semilla para luego crear un número aleatorio
            hiprandState* d_state;
            hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));
            unsigned int ale = generate_seed();
            eliminar7oMas <<<num_blocks, block_size >> > (d_matriz, n, m, d_vector, fila, columna, ale, d_state,lim_sup,lim_inf);
            break;
        }


    // Copiar la matriz resultante de la GPU a la CPU
    hipMemcpy(matriz, d_matriz, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(vector, d_vector, n * m * sizeof(int), hipMemcpyDeviceToHost);
    // Liberar la memoria de la GPU
    hipFree(d_matriz);
    hipFree(d_vector);
}


//devuelve el minimo entre a y b
int minimo(int a, int b) {
    return (a < b) ? a : b;
}

//Calcula las mejores características para una mejor optimizacion
void mejoresCaracteristicas(int n, int m) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // Calculamos el número máximo de bloques por multiprocesador
    int max_blocks_per_sm = deviceProp.maxBlocksPerMultiProcessor;

    // Calculamos el número máximo de hilos por multiprocesador
    int max_threads_per_sm = deviceProp.maxThreadsPerMultiProcessor;

    // Calculamos el número de hilos por bloque
    int hilos_por_bloque_x = minimo(m, max_threads_per_sm);
    int hilos_por_bloque_y = minimo(n, max_threads_per_sm);

    // Calculamos el número de bloques por dimensión
    int bloques_por_dim_x = ceil((float)m / hilos_por_bloque_x);
    int bloques_por_dim_y = ceil((float)n / hilos_por_bloque_y);

    // Limitamos el número de bloques a lanzar por multiprocesador
    int bloques_por_sm = max_blocks_per_sm / (bloques_por_dim_x * bloques_por_dim_y);

    // Calculamos el número de bloques a lanzar
    int num_bloques = bloques_por_dim_x * bloques_por_dim_y;
    if (hilos_por_bloque_x * hilos_por_bloque_y > max_threads_per_sm) {
        num_bloques = minimo(num_bloques, bloques_por_sm * deviceProp.multiProcessorCount);
        hilos_por_bloque_x = minimo(m, max_threads_per_sm / hilos_por_bloque_y);
        hilos_por_bloque_y = minimo(n, max_threads_per_sm / hilos_por_bloque_x);
        bloques_por_dim_x = ceil((float)m / hilos_por_bloque_x);
        bloques_por_dim_y = ceil((float)n / hilos_por_bloque_y);
    }

    // Asignamos los valores finales a las variables globales
    BLOQUES_GRID_X = bloques_por_dim_x;
    BLOQUES_GRID_Y = bloques_por_dim_y;
    HILOS_BLOQUE_X = hilos_por_bloque_x;
    HILOS_BLOQUE_Y = hilos_por_bloque_y;

    printf("\n(Hilos/Bloque).x: %d\n", HILOS_BLOQUE_X);
    printf("\n(Hilos/Bloque).y: %d\n", HILOS_BLOQUE_Y);
    printf("\n(Bloques/Grid).x: %d\n", BLOQUES_GRID_X);
    printf("\n(Bloques/Grid).y: %d\n", BLOQUES_GRID_Y);
}

//imprimir la matriz
void imprimir(int* matriz, int n, int m) {
    char str[10];
    printf("\n");
    for (int i = -1; i < n; i++) {
        for (int j = -1; j < m; j++) {
            if (i == -1 && j == -1) {
                printf("_");
            }
            else if (i == -1 && j >= 0) {
                printf("|");
                printf("\x1b[4m%d\x1b[0m", j);
                printf("|");
                printf(" ");
            }
            else if (i != -1 && j == -1) {
                printf("\x1b[4m%d\x1b[0m", i);
                printf("|");
            }
            else if (matriz[i * m + j] == -1) {
                printf("    "); //elemento borrado, no se pone 
            }
            else if (matriz[i * m + j] == 1) {
                sprintf(str, "%d", matriz[i * m + j]);
                printf("\x1b[36m%s   \x1b[0m", str);//caramelo azul, 1
            }
            else if (matriz[i * m + j] == 2) {
                sprintf(str, "%d", matriz[i * m + j]);
                printf("\x1b[31m%s   \x1b[0m", str);//caramelo rojo, 2
            }
            else if (matriz[i * m + j] == 3) {
                sprintf(str, "%d", matriz[i * m + j]);
                printf("\x1b[38;5;226m%s   \x1b[0m", str); //caramelo naranja, 3
            }
            else if (matriz[i * m + j] == 4) {
                sprintf(str, "%d", matriz[i * m + j]);
                printf("\x1b[32m%s   \x1b[0m", str);// caramelo verde, 4
            }
            else if (matriz[i * m + j] == 5) {
                sprintf(str, "%d", matriz[i * m + j]);
                printf("\x1b[38;5;130m%s   \x1b[0m", str);//caramelo marron, 5
            }
            else if (matriz[i * m + j] == 6) {
                sprintf(str, "%d", matriz[i * m + j]);
                printf("\x1b[38;5;165m%s   \x1b[0m", str);//caramelo lila, 6
            }
            else if (matriz[i * m + j] == 7) {
                printf("\x1b[23;5;214mB   \x1b[0m");//Bomba
            }
            else if (matriz[i * m + j] == 8) {
                printf("\x1b[23;5;214mT   \x1b[0m");//TNT
            }
            else if (matriz[i * m + j] == 9) {
                printf("\x1b[36;5;214mR1  \x1b[0m");//R1
            }
            else if (matriz[i * m + j] == 10) {
                printf("\x1b[31;5;214mR2  \x1b[0m");//R2
            }
            else if (matriz[i * m + j] == 11) {
                printf("\x1b[38;5;226;5;214mR3  \x1b[0m");//R3
            }
            else if (matriz[i * m + j] == 12) {
                printf("\x1b[32;5;214mR4  \x1b[0m");//R4
            }
            else if (matriz[i * m + j] == 13) {
                printf("\x1b[38;5;130;5;214mR5  \x1b[0m");//R5
            }
            else if (matriz[i * m + j] == 14) {
                printf("\x1b[38;5;165;5;214mR6  \x1b[0m");//R6
            }
        }
        printf("\n");
    }
}


int main()
{
    srand(time(NULL));
    char tecla;
    int modo; //automático o manual
    int dificultad; //dificultad del juego
    int n; // número de filas
    int m; // número de columnas

  
    //Obtencion de valores de modo de juego
    printf("Bienvenido a Cundio Crack\n");
    printf("Introduce el modo de juego con el que quieres jugar: \n 1. Automatico \n 2. Manual \n");
    scanf("%d", &modo);
    printf("Introduce la dificultad con la que quieres jugar: \n 1. Facil \n 2. Normal \n");
    scanf("%d", &dificultad);
    printf("Introduce el numero de filas que quieres que tenga el tablero: \n");
    scanf("%d", &n);
    printf("Introduce el numero de columnas que quieres que tenga el tablero: \n");
    scanf("%d", &m);

    mejoresCaracteristicas(n, m);

    int lim_inf = 1; // valor mínimo
    int lim_sup = 6; // valor máximo
    if (dificultad == 1) {
         lim_sup = 4; // valor máximo
    }
    
    int* mat = (int*)malloc(n * m * sizeof(int)); // matriz aleatoria
    int* posicionesVistas = (int*)malloc(n * m * sizeof(int)); //Vector donde se guardan posiciones adyacentes
    crear_vector(posicionesVistas, n, m);//Inicializa el vector
    crear_matriz_aleatoria(mat, n, m, lim_inf, lim_sup);//Inicializacion de la matriz

    dim3 block_size(HILOS_BLOQUE_X, HILOS_BLOQUE_Y);
    dim3 num_blocks(BLOQUES_GRID_X, BLOQUES_GRID_Y);

    int colum=-1;
    int fila=-1;
    int x = 0;
    //Parte3
    printf("\n\x1b[31;5;214m%d VIDAS RESTANTES\x1b[0m\n", vidas);
    imprimir(mat, n, m);
    printf("\n");


    while (vidas > 0) {
        
        do {
            if (modo == 2) {
                //obtencion de filas y columnas por parte del usuario
                printf("\nIntroduce la fila del caramelo que quieres comprobar\n");
                scanf("%d", &fila);
                printf("Introduce la columna del caramelo que quieres comprobar\n");
                scanf("%d", &colum);
            }
            else { 
                //obtencion de filas y columna de manera aleatoria
                x++;
                srand(time(NULL) + x);
                colum =rand()%m; 
                x++;
                srand(time(NULL) + x);
                fila = rand()%n;

                printf("\nFila escogida: %d\n",fila);
                printf("Columna escogida: %d\n", colum);

                printf("\nPULSA ENTER PARA CONTINUAR\n");
                fflush(stdin);
                tecla = getchar();

            }
            if (colum > m - 1 || fila > n - 1 || colum < 0 || fila < 0) {
                printf("\nCOORDENADAS NO VALIDAS, introduce unas coordenadas posibles\n\n");
            }
        } while (colum > m-1 || fila > n-1 || colum < 0 || fila < 0);//comprobacion de que las filas introducidas no son validas
        
        int elemento = mat[fila * m + colum]; //caramelo en las coordenadas indicadas
        ver_candy(mat, n, m, colum, fila, posicionesVistas,elemento); //obtenemos un vector con todas los caramelos adyacentes al seleccionado, incluido el propio caramelo 
        printf("\n");

        eliminar_elementos(mat, n, m, posicionesVistas,fila,colum,lim_sup,lim_inf);//Eliminacion de la posicion seleccionada y sus adyacentesç
        imprimir(mat, n, m);
        caer_caramelos_host(mat, n, m);//caida de los caramelos que tengan elementos eliminados por debajo
        imprimir(mat, n, m);
        rellenar_huecos_host(mat, n, m, lim_inf, lim_sup);//donde haya elementos eliminados se ponen nuevos caramelos aleatorios

        if (vidas == 0)printf("\n\x1b[31;5;214mTE HAS QUEDADO SIN VIDAS\x1b[0m\n");
        else if (vidas > 1)printf("\n\x1b[31;5;214m%d VIDAS RESTANTES\x1b[0m\n", vidas);
        else printf("\n\x1b[31;5;214m%d VIDA RESTANTE\x1b[0m\n", vidas);

        imprimir(mat, n, m);
    }

    printf("\n-----Trabajo realizado por Jaime Diez Buendia y Eduardo Garcia Huerta-----\n");
    //liberacion del puntero 
    free(mat);

    //fin del programa
    return 0;
}