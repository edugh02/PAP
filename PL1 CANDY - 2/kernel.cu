#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>


int vidas = 5;
 //y filas, x columnas --> idy*columnas + idx ( idx es todos los .x)


// Esta función genera una semilla aleatoria basada en la hora actual.
unsigned int generate_seed() {
    time_t t;
    time(&t);
    return (unsigned int)t % 100000;
}

//Comprueba si una posicion esta en el vector
bool esta_en_vector(int* vector, int m, int n, int pos) {
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == pos) {
            return true;
        }
    }
    return false;
}

//retorna la primera poscion donde haya un -1
int primer_vacio(int* vector, int n, int m) {
    int x = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] == -1) {
            x = i;
            i = n * m;
        }
    }
    return x;
}


void ver_candy(int* mat, int n, int m,int colum,int fila, int* vector,int elemento) {
    int caramelo = fila * m + colum;//posicion en la matriz de las coordenadas

    if (!esta_en_vector(vector,m,n,caramelo)&& mat[caramelo]==elemento) {
        //comprobamos que la posicion no ha sido ya insertada
        int pos=primer_vacio(vector,n,m);
        vector[pos] = caramelo;
        //insertamos en la primera posicion que se encuentre vacia del vector, la posicion del caramelo
        if (fila != 0) {//Adyacente de arriba
            ver_candy(mat, n, m, colum, fila - 1, vector,elemento);
        }
        if (fila != n - 1) {//Adyacente de abajo
            ver_candy(mat, n, m, colum, fila + 1, vector,elemento);
        }
        if (colum != 0) {//Adyacente de la izquierda
            ver_candy(mat, n, m, colum - 1, fila, vector,elemento);
        }
        if (colum != m - 1) {//Adyacente de la derecha
            ver_candy(mat, n, m, colum + 1, fila, vector,elemento);
        }
        
    }    
}

//obtencion de cuantas posiciones han sido encontradas para ser borradas
int cuantas_posiciones(int* vector, int n, int m) {
    int contador = 0;
    for (int i = 0; i < n * m; ++i) {
        if (vector[i] != -1) {
            ++contador;
        }
    }
    return contador;
}

// Esta función genera una matriz aleatoria de números enteros entre "lim_inf" y "lim_sup".
__global__ void matriz_aleatoria(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    // Verificar si el hilo se encuentra dentro de los límites de la matriz
    if (idx < m && idy < n) {
        // Inicializar el generador de números aleatorios
        hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
        // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
        int val = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;
        // Asignar el valor aleatorio a la matriz
        mat[idy * m + idx] = val;
    }
}

__global__ void rellenar_huecos(int* mat, int n, int m, int lim_inf, int lim_sup, unsigned int ale, hiprandState* state) {
        // Calcular las coordenadas x e y del hilo
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        int idy = threadIdx.y + blockDim.y * blockIdx.y;

        // Verificar si el hilo se encuentra dentro de los límites de la matriz
        if (idx < m && idy < n && mat[idy * m + idx] == -1) {
            // Inicializar el generador de números aleatorios
            hiprand_init(ale,idy * m + idx, 0, &state[idy * m + idx]);
            // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
            int val = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;
            
            // Asignar el valor aleatorio al hueco
            mat[idy * m + idx] = val;
            
        }
}

__global__ void eliminar_iguales_juntos(int* mat, int n, int m,int* vector) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos;

    for (int i = 0; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        } 
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        //el elemento deja de ser analizado
        vector[pos] = -1;
    }
}
__global__ void eliminar5(int* mat, int n, int m, int* vector,int fila,int columna) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos=fila*m+columna;
    
    if (vector[0] == idy * m + idx) {
        mat[vector[0]] = 7;//Se pone una bomba en la posicion
        vector[0] = -1;
    }

    for (int i = 1; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}

__global__ void eliminar6(int* mat, int n, int m, int* vector,int fila,int columna) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos = fila * m + columna;
    
    if (vector[0] == idy * m + idx) {
        mat[fila * m + columna] = 8;//se pone una TNT en la posicion 
        vector[0] = -1;
    }

    for (int i = 1; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}

__global__ void eliminar7oMas(int* mat, int n, int m, int* vector,int fila,int columna, unsigned int ale, hiprandState* state,int lim_sup,int lim_inf) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    bool centinela = false;
    int pos;
    hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
    // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
    int val_ale = hiprand(&state[idy * m + idx]) % lim_sup + lim_inf;
    //generar numero aleatorio que indicará que tipo de r se forma
    
    if (vector[0] == idy * m + idx) {//9-->R1 10-->R2 11-->R3 21-->R4 13-->R5 14-->R6
        if (val_ale == 1) {//En la posicion se pone un R1
            mat[idy * m + idx] = 9;
        }
        else if (val_ale == 2) {//En la posicion se pone un R2
            mat[idy * m + idx] = 10;
        }
        else if (val_ale == 3) {//En la posicion se pone un R3
            mat[idy * m + idx] = 11;
        }
        else if (val_ale == 4) {//En la posicion se pone un R4
            mat[idy * m + idx] = 12;
        }
        else if (val_ale == 5) {//En la posicion se pone un R5
            mat[idy * m + idx] = 13;
        }
        else {//En la posicion se pone un R6
            mat[idy * m + idx] = 14;
        }
        vector[0] = -1;
    }

    for (int i = 1; i < n * m; i++) {
        if (vector[i] == idy * m + idx) {
            centinela = true; // El número está presente en el vector
            pos = i;
        }
    }

    // Verificar si el hilo se encuentra dentro de los límites de la matriz y coincide con una posición que hay que eliminar
    if (centinela) {
        mat[idy * m + idx] = -1;
        vector[pos] = -1;
    }
}

__global__  void caer_caramelos(int* matriz, int n, int m) {
    // Calcular las coordenadas x e y del hilo
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;


    // Contar los elementos -1 debajo del hilo
    int num_minus_1 = 0;
    for (int i = idy; i < n; ++i) {
        if (matriz[i * m + idx] == -1) {
            num_minus_1++;
        }
    }
    __syncthreads();
    // Buscar el primer elemento -1 debajo del hilo y intercambiarlo
    if (num_minus_1 > 0 && matriz[idy * m + idx] != -1) {
        int aux = matriz[idy * m + idx];
        matriz[idy * m + idx] = -1;
        matriz[(idy + num_minus_1) * m + idx] = aux;
    }
}



void caer_caramelos_host(int* matriz, int n, int m) {
    //Creacion de puntero para la GPU
    int* d_matriz;

    //Reservamos memoria 
    int size = n * m * sizeof(int);
    hipMalloc((void**)&d_matriz, size);

    //copiamos del host al device
    hipMemcpy(d_matriz, matriz, size, hipMemcpyHostToDevice);

    //TODO
    // Configurar la cantidad de hilos por bloque y la cantidad de bloques por cuadrícula
    dim3 tamBloque(m, n);
    dim3 tamCuadricula(1,1);

    // Llamar al kernel caer_caramelos
    caer_caramelos <<<tamCuadricula, tamBloque >> > (d_matriz, n, m);

    // Copiar la matriz resultante de la GPU al host
    hipMemcpy(matriz, d_matriz, size, hipMemcpyDeviceToHost);

    // Liberar memoria de la GPU
    hipFree(d_matriz);
}


__global__ void rellenar(int* vec, int n,int m) {
    //rellenar el vector con -1

    //posicion del hilo correspondiente al vector
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    if (idy < n && idx < m) {
        vec[idy*m+idx] = -1;
    }
}

void crear_vector(int* posicionesVistas, int n, int m) {
    //creacion de un puntero para la GPU
    int* d_v;
    //Reservamos memoria
    hipMalloc((void**)&d_v, n * m * sizeof(int));

    // Definir la configuración del kernel 
    //TODO
    dim3 threadsPerBlock (m,n);
    dim3 blocksPerGrid (1,1) ;

    // Llamar al kernel
    rellenar <<<blocksPerGrid, threadsPerBlock >> > (d_v, n, m);

    //Copiamos en el host el vector creado por el kernel
    hipMemcpy(posicionesVistas, d_v, n * m * sizeof(int), hipMemcpyDeviceToHost);
    //liberamos el puntero
    hipFree(d_v);
}



void crear_matriz_aleatoria(int* mat, int n, int m, int lim_inf, int lim_sup) {
    //creacion de un puntero para la GPU
    int* d_mat;
    //reservamos espacio en la memoria
    hipMalloc((void**)&d_mat, n * m * sizeof(int));

    //puntero que ayudara a la obtencion de numeros aleatorios dentro del Kernel
    hiprandState* d_state;
    //reservamos espacio en la memoria
    hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));

    //obtencion de una semilla que ayudara a la creacion de numeros aleatorios
    unsigned int ale = generate_seed();

    //TODO
    dim3 block_size(m,n);
    dim3 num_blocks(1,1);
    matriz_aleatoria <<<num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);
    
    //Copiamos en el host el resultado obtenido en el device
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    //liberacion de punteros
    hipFree(d_mat);
    hipFree(d_state);
}

void rellenar_huecos_host(int* mat, int n, int m, int lim_inf, int lim_sup) {
    //creacion de un puntero para la GPU
    int* d_mat;

    //Reservamos memoria 
    hipMalloc((void**)&d_mat, n * m * sizeof(int));

    //Copiamos del host al device 
    hipMemcpy(d_mat, mat, n * m * sizeof(int), hipMemcpyHostToDevice);

    //Creacion de un puntero que ayudara a la creacion de numeros aleatorios
    hiprandState* d_state;
    hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));

    unsigned int ale = generate_seed();

    //TODO
    dim3 block_size(m, n);
    dim3 num_blocks(1,1);
    rellenar_huecos <<<num_blocks, block_size >> > (d_mat, n, m, lim_inf, lim_sup, ale, d_state);

    //Copiamos del device al host
    hipMemcpy(mat, d_mat, n * m * sizeof(int), hipMemcpyDeviceToHost);

    //liberamos punteros
    hipFree(d_mat);
    hipFree(d_state);
}


__global__ void explotarBomba(int* mat, int n, int m, int fila, int columna, unsigned int ale, hiprandState* state,int* vector) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x; //coordenada de la fila del hilo
    int idy = threadIdx.y + blockDim.y * blockIdx.y; //coordenada de la columna del hilo
                                                                        //TODO falta hacer la concatenacion
    hiprand_init(ale, idy * m + idx, 0, &state[idy * m + idx]);
    // Generar un número aleatorio entero entre "lim_inf" y "lim_sup"
    int tipo = hiprand(&state[idy * m + idx]) % 1 + 0;

    if (tipo == 0) {//Eliminar la columna entera
        if (idx == columna) {
            mat[idy * m + idx] = -1;
        }
    }
    else {//Eliminar la fila entera
        if (idy == fila) {
            mat[idy * m + idx] = -1;
        }
    }
    //el elemento deja de ser analizado
    vector[0] = -1;
}

__global__ void explotarTNT(int* mat, int n, int m, int fila, int columna,int* vector) { 
    int idx = threadIdx.x + blockDim.x * blockIdx.x; //coordenada de la fila del hilo
    int idy = threadIdx.y + blockDim.y * blockIdx.y; //coordenada de la columna del hilo
                                                                            //TODO falta hacer la concatenacion
    mat[fila * m + columna] = -1;

    if (fila != 0 && idy==fila-1) {//comprobamos que no estamos en la primera fila
        if (columna != 0 && idx== columna-1) { //comprobamos que no estamos en la primera columna para borrar el elemento de la izq
            mat[idy * m + idx] = -1;
        }
        else if (columna != m && idx == columna + 1) { //comprobamos que no estamos en la ultima columna para borrar el elem de la der
            mat[idy * m + idx] = -1;
        }
        else if (idx == columna) {
            //comprobamos que que nosea la primera fila para borrar el de arriba
            if (0 < mat[idy * m + idx] && mat[idy * m + idx] < 7) {
                mat[idy * m + idx] = -1;
            }
        }
    }
    else if (fila != n && idy == fila+1) { //comprobamos que la fila no es la última
        if (columna != 0 && idx == columna-1) {//comprobamos que no estamos en la primera columna para borrar el elemento de la izq
            mat[idy * m + idx] = -1;
        }
        else if (columna != m && idx == columna + 1) { //comprobamos que no estamos en la ultima columna para borrar el elem de la der
            mat[idy * m + idx] = -1;
        }
        else if (idx == columna) {
            //comprobamos que que nosea la primera fila para borrar el de arriba
            if (0 < mat[idy * m + idx] && mat[idy * m + idx] < 7) {
                mat[idy * m + idx] = -1;
            }
        }
    }
    //para borrar el elemento de la izq, comprobamos que no estamos en la primera columna
    if (columna != 0 && idy==fila && idx == columna-1) {
        mat[idy * m + idx] = -1;
    }
    //para borrar el elemento de la der, comprobamos que no estamos en la ultima columna
    else if (columna != m && idy == fila && idx == columna + 1) {
        mat[idy * m + idx] = -1;
    }
    //el elemento deja de ser analizado
    vector[0] = -1;
}

__global__ void explotarRx(int* matriz, int n, int m, int fila, int columna, int tipo,int *vector) { 
    int idx = threadIdx.x + blockDim.x * blockIdx.x;//coordenada de la fila del hilo
    int idy = threadIdx.y + blockDim.y * blockIdx.y;//coordenada de la columna del hilo
    int pos = idy * m + idx;//posicion del hilo
    matriz[fila*m+columna] = -1;
    switch (tipo) {
        case 9://R1-->eliminacion de todos los 1 de la matriz
            if (matriz[pos] == 1) {
                matriz[pos]=-1;
            }
            break;
        case 10://R2-->eliminacion de todos los 2 de la matriz
            if (matriz[pos] == 2) {
                matriz[pos] = -1;
            }
            break;
        case 11://R3-->eliminacion de todos los 3 de la matriz
            if (matriz[pos] == 3) {
                matriz[pos] = -1;
            }
            break;
        case 12://R4-->eliminacion de todos los 4 de la matriz
            if (matriz[pos] == 4) {
                matriz[pos] = -1;
            }
            break;
        case 13://R5-->eliminacion de todos los 5 de la matriz
            if (matriz[pos] == 5) {
                matriz[pos] = -1;
            }
            break;
        case 14://R6-->eliminacion de todos los 6 de la matriz
            if (matriz[pos] == 6) {
                matriz[pos] = -1;
            }
            break;
        default:
            break;
    }
    //el elemento deja de ser analizado
    vector[0] = -1;
}

__global__ void eliminar1(int* mat, int* vector, int n, int m, int fila, int columna) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;//coordenada de la fila del hilo
    int idy = threadIdx.y + blockDim.y * blockIdx.y;//coordenada de la columna del hilo
    int pos = idy * m + idx;//posicion del hilo
    if (pos == fila * m + columna) {
        vector[0] = -1;
        mat[pos] = -1;
    }
}

void eliminar_elementos(int* matriz, int n, int m, int* vector, int fila, int columna,int lim_sup, int lim_inf) {                       //AQUI HAY Q LLAMAR BIEN A LOS MÉTODOS DEL KERNEL
    //creacion de punteros para la GPU
    int* d_matriz;
    int* d_vector;

    int tamVector = n * m;
    srand(time(NULL));

    // Alocar memoria para la matriz y el vector en la GPU
    hipMalloc((void**)&d_matriz, n * m * sizeof(int));
    hipMalloc((void**)&d_vector, tamVector * sizeof(int));

    // Copiar la matriz y el vector de la CPU a la GPU
    hipMemcpy(d_matriz, matriz, n * m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, vector, tamVector * sizeof(int), hipMemcpyHostToDevice);

    // Definir la configuración del kernel
    dim3 blockSize(m, n);
    dim3 gridSize(1,1);

    // Llamar al kernel
    switch (cuantas_posiciones(vector, n, m)) {
        //Comprobamos cuantas posiciones adyacentes hay
        case 1: 
            //No hay posciones adyacentes
            if (matriz[vector[0]] == 1 || matriz[vector[0]] == 2 || matriz[vector[0]] == 3 || matriz[vector[0]] == 4 || matriz[vector[0]] == 5 || matriz[vector[0]] == 6) {
                vidas--;
                eliminar1 << <gridSize, blockSize >> > (d_matriz, d_vector, n, m, fila, columna);
            }
            else if (matriz[vector[0]] == 7) {
                //BOMBA
                //generamos la semilla para luego crear un número aleatorio
                hiprandState* d_state;
                hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));
                unsigned int ale = generate_seed();
                explotarBomba <<<gridSize, blockSize >> > (d_matriz, n, m, fila, columna, ale, d_state,d_vector);
            }
            else if (matriz[vector[0]] == 8) {
                //TNT
                explotarTNT <<<gridSize, blockSize >> > (d_matriz, n, m, fila, columna,d_vector);
            }
            else if (matriz[vector[0]] > 8) {
                //Rx
                int tipo = matriz[vector[0]];
                explotarRx <<<gridSize, blockSize >> > (d_matriz, n, m, fila, columna, tipo,d_vector);
            }
            break;
        //Hay 1 caramelo adyacente
        case 2:
            eliminar_iguales_juntos <<<gridSize, blockSize >> > (d_matriz, n, m, d_vector);
            break;
        //Hay 2 caramelos adyacentes
        case 3:
            eliminar_iguales_juntos <<<gridSize, blockSize >> > (d_matriz, n, m, d_vector);
            break;
        //Hay 3 caramelos adyacentes
        case 4:
            eliminar_iguales_juntos <<<gridSize, blockSize >> > (d_matriz, n, m, d_vector);
            break;
        //Hay 4 caramelos adyacentes (se boorran 5 elementos-->bomba)
        case 5:
            //Kernel sustituir el elemento de la posición por un B y borrar el resto
            eliminar5 <<<gridSize, blockSize >> > (d_matriz, n, m, d_vector, fila, columna);
            break;
        //Hay 5 caramelos adyacentes (se boorran 6 elementos-->TNT)
        case 6:
            //Kernel sustituir el elemento de la posición por un TNT y borrar el resto
            eliminar6 <<<gridSize, blockSize >> > (d_matriz, n, m, d_vector, fila, columna);
            break;
        //Hay 6 o mas caramelos adyacentes (se boorran 7 o mas elementos-->Rx)
        default:
            //Kernel sustituir el elemento de la posición por un Rx y borrar el resto
            int tipo = rand() % 6+1;
            //generamos la semilla para luego crear un número aleatorio
            hiprandState* d_state;
            hipMalloc((void**)&d_state, n * m * sizeof(hiprandState));
            unsigned int ale = generate_seed();
            eliminar7oMas <<<gridSize, blockSize >> > (d_matriz, n, m, d_vector, fila, columna, ale, d_state,lim_sup,lim_inf);
            break;
        }


    // Copiar la matriz resultante de la GPU a la CPU
    hipMemcpy(matriz, d_matriz, n * m * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(vector, d_vector, n * m * sizeof(int), hipMemcpyDeviceToHost);
    // Liberar la memoria de la GPU
    hipFree(d_matriz);
    hipFree(d_vector);
}

//imprimir la matriz
void imprimir(int* matriz, int n, int m) {
    printf("\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            if (matriz[i * m + j] == -1) {
                printf("    "); //elemento borrado, no se pone 
            }else if (matriz[i * m + j]==7) {
                printf("B   ");//bomba
            }else if (matriz[i * m + j] == 8) {
                printf("T   ");//TNT
            }
            else if (matriz[i * m + j] == 9) {
                printf("R1  ");//R1
            }
            else if (matriz[i * m + j] == 10) {
                printf("R2  ");//R2
            }
            else if (matriz[i * m + j] == 11) {
                printf("R3  ");//R3
            }
            else if (matriz[i * m + j] == 12) {
                printf("R4  ");//R4
            }
            else if (matriz[i * m + j] == 13) {
                printf("R5  ");//R5
            }
            else if (matriz[i * m + j] == 14) {
                printf("R6  ");//R6
            }
            else {
                printf("%d   ", matriz[i * m + j]);//elementos normales de la matriz
            }
        }
        printf("\n");
    }
}


int main()
{
    srand(time(NULL));
    char tecla;
    int modo; //automático o manual
    int dificultad; //dificultad del juego
    int n; // número de filas
    int m; // número de columnas

    //Obtencion de valores de modo de juego
    printf("Bienvenido a Cundio Crack\n");
    printf("Introduce el modo de juego con el que quieres jugar: \n 1. Automatico \n 2. Manual \n");
    scanf("%d", &modo);
    printf("Introduce la dificultad con la que quieres jugar: \n 1. Facil \n 2. Normal \n");
    scanf("%d", &dificultad);
    printf("Introduce el numero de filas que quieres que tenga el tablero: \n");
    scanf("%d", &n);
    printf("Introduce el numero de columnas que quieres que tenga el tablero: \n");
    scanf("%d", &m);

    int lim_inf = 1; // valor mínimo
    int lim_sup = 6; // valor máximo
    if (dificultad == 1) {
         lim_sup = 4; // valor máximo
    }
    
    int* mat = (int*)malloc(n * m * sizeof(int)); // matriz aleatoria
    int* posicionesVistas = (int*)malloc(n * m * sizeof(int)); //Vector donde se guardan posiciones adyacentes
    crear_vector(posicionesVistas, n, m);//Inicializa el vector
    crear_matriz_aleatoria(mat, n, m, lim_inf, lim_sup);//Inicializacion de la matriz

    //TODO                                                                                 
    dim3 dimBlock(m,n);                             //<--ESTO HAY QUE VER COMO HACERLO EFICIENTE
    dim3 dimGrid(1,1);

    int colum=-1;
    int fila=-1;
    int x = 0;

    printf("\n%d VIDAS RESTANTES\n", vidas);
    imprimir(mat, n, m);
    printf("\n");


    while (vidas > 0) {
        
        do {
            if (modo == 2) {
                //obtencion de filas y columnas por parte del usuario
                printf("\nIntroduce la fila del caramelo que quieres comprobar\n");
                scanf("%d", &fila);
                printf("Introduce la columna del caramelo que quieres comprobar\n");
                scanf("%d", &colum);
            }
            else { 
                //obtencion de filas y columna de manera aleatoria
                x++;
                srand(time(NULL) + x);
                colum =rand()%m; 
                x++;
                srand(time(NULL) + x);
                fila = rand()%n;

                printf("\nFila escogida: %d\n",fila);
                printf("Columna escogida: %d\n", colum);

                printf("\nPULSA ENTER PARA CONTINUAR\n");
                fflush(stdin);
                tecla = getchar();

            }
            if (colum > m - 1 || fila > n - 1 || colum < 0 || fila < 0) {
                printf("\nCOORDENADAS NO VALIDAS, introduce unas coordenadas posibles\n\n");
            }
        } while (colum > m-1 || fila > n-1 || colum < 0 || fila < 0);//comprobacion de que las filas introducidas no son validas
        
        int elemento = mat[fila * m + colum]; //caramelo en las coordenadas indicadas
        ver_candy(mat, n, m, colum, fila, posicionesVistas,elemento); //obtenemos un vector con todas los caramelos adyacentes al seleccionado, incluido el propio caramelo 

        printf("\n");
        eliminar_elementos(mat, n, m, posicionesVistas,fila,colum,lim_sup,lim_inf);//Eliminacion de la posicion seleccionada y sus adyacentesç
        imprimir(mat, n, m);
        caer_caramelos_host(mat, n, m);//caida de los caramelos que tengan elementos eliminados por debajo
        imprimir(mat, n, m);
        rellenar_huecos_host(mat, n, m, lim_inf, lim_sup);//donde haya elementos eliminados se ponen nuevos caramelos aleatorios

        printf("\n%d VIDAS RESTANTES\n", vidas);
        imprimir(mat, n, m);
    }

    printf("\nFIN DE JUEGO\n");
    //liberacion del puntero 
    free(mat);

    //fin del programa
    return 0;
}